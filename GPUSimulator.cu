#include <iostream>
#include "src/circuit_model/circuit.h"
#include "src/vcd_reader.h"
#include "src/utils.h"
#include "src/simulation_result.h"
#include "simulator/simulator.h"
#include "simulator/memory_manager.h"
#include "simulator/module_registry.h"


using namespace std;


void print_usage() {
    cout << "| Usage: GPUSimulator.cu.py "
            "<intermediate_representation.file> "
            "<input.vcd> "
            "<SAIF_or_VCD_flag> "
            "<dumpon_time> "
            "<dumpoff_time> "
            "[SAIF_or_output_VCD.saif.vcd]" << endl;
}


bool arguments_valid(int argc, char* argv[1]) {
    if (argc != 7) {
        cerr << "| Error: Wrong number of arguments" << endl;
        print_usage();
        return false;
    }
    string output_flag = string(argv[3]);
    if (output_flag != "SAIF" and output_flag != "VCD") {
        cerr << "| Error: The third argument should be either 'SAIF' or 'VCD'" << endl;
        return false;
    }
    return true;
}

void check_cuda_device() {
    int device;
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess) cerr << "| Error: " << hipGetErrorString(err) << endl;
}

int main(int argc, char* argv[]) {
    check_cuda_device();

    if (not arguments_valid(argc, argv))
        return -1;

    char* inter_repr_file = argv[1];
    char* input_vcd_file = argv[2];
    string output_flag = string(argv[3]);
    Timestamp dumpon_time = atoll(argv[4]);
    Timestamp dumpoff_time = atoll(argv[5]);
    char* output_file_name = argv[6];

    ifstream fin_intermediate = ifstream(inter_repr_file);
    if (!fin_intermediate) throw runtime_error("Bad intermediate file.");
    ModuleRegistry module_registry;
    module_registry.read_file(fin_intermediate);
    module_registry.summary();

    BusManager bus_manager;
    Circuit circuit(module_registry);
    VCDReader vcd_reader(input_vcd_file);
    InputInfo input_info = vcd_reader.read_input_info();
    input_info.summary();

    circuit.read_intermediate_file(fin_intermediate, input_info.timescale, bus_manager);
    fin_intermediate.close();
    vcd_reader.read_input_waveforms(circuit);
    vcd_reader.summary();
    circuit.summary();

    Simulator simulator(circuit);
    simulator.run();

    SimulationResult* simulation_result;
    if (output_flag == "SAIF") {
        simulation_result = new SAIFResult(
            circuit.wires,
            input_info.scopes,
            input_info.timescale_pair,
            dumpon_time, dumpoff_time,
            bus_manager
        );
    } else if (output_flag == "VCD") {
        simulation_result = new VCDResult(
            circuit.wires,
            input_info.scopes,
            input_info.timescale_pair,
            dumpon_time, dumpoff_time,
            bus_manager
        );
    }

    simulation_result->write(output_file_name);
}
