#include <iostream>
#include <string>

#include "utils.h"

using namespace std;


double get_timescale(int num, const string& unit) {
    double unit_num;
    if (unit == "ms") {
        unit_num = 1e-6;
    } else if (unit == "ns") {
        unit_num = 1e-9;
    } else if (unit == "ps") {
        unit_num = 1e-12;
    } else throw runtime_error("Unrecognized timescale unit: " + unit + "\n");

    return num * unit_num;
}

__host__ __device__ char get_edge_type(const char& v1, const char& v2) {
    if (v2 == '1' or v1 == '0') return '+';
    if (v2 == '0' or v1 == '1') return '-';
    return 'x';
}

void cudaErrorCheck(hipError_t status) {
    if (status != hipSuccess) throw runtime_error(hipGetErrorString(status));
}
