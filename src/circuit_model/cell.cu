#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <utility>
#include "cell.h"
#include "utils.h"

using namespace std;


Cell::Cell(
    const ModuleSpec* module_spec,
    const StdCellDeclare* declare,
    const WireMap<Wire>& pin_specs,
    string name
) : module_spec(module_spec), name(std::move(name))
{
    num_args = declare->num_input + declare->num_output;
    build_wire_map(declare, pin_specs);
}

void Cell::set_paths() {
    vector<char> edge_types;
    vector<unsigned int> input_indices, output_indices;
    vector<int> rising_delays, falling_delays;

    for (const auto& path : sdf_paths) {
        input_indices.push_back(path.in);
        output_indices.push_back(path.out);
        edge_types.push_back(path.edge_type);
        rising_delays.push_back(path.rising_delay);
        falling_delays.push_back(path.falling_delay);
    }

    const auto& num_rows = sdf_paths.size();
    host_sdf_spec.num_rows = num_rows;
    hipMalloc((void**) &host_sdf_spec.edge_type, sizeof(char) * num_rows);
    hipMalloc((void**) &host_sdf_spec.input_index, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.output_index, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.rising_delay, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.falling_delay, sizeof(int) * num_rows);
    hipMemcpy(host_sdf_spec.edge_type, edge_types.data(), sizeof(char) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.input_index, input_indices.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.output_index, output_indices.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.rising_delay, rising_delays.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.falling_delay, falling_delays.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);

    hipMalloc((void**) &sdf_spec, sizeof(SDFSpec));
    hipMemcpy(sdf_spec, &host_sdf_spec, sizeof(SDFSpec), hipMemcpyHostToDevice);
}

void Cell::build_wire_map(
    const StdCellDeclare* declare,
    const WireMap<Wire>& pin_specs
) {
    if (num_args > MAX_NUM_MODULE_ARGS) {
        throw runtime_error("Too many module args (" + to_string(num_args) + ")\n");
    }
    for (unsigned int arg = 0; arg < declare->num_input; ++arg) {
        auto* wire_ptr = pin_specs.get(arg);
        if (wire_ptr == nullptr) continue;
        auto* wrapped_wire = new InputWire(wire_ptr);
        wire_map.set(arg, wrapped_wire); input_wires.push_back(wrapped_wire);
    }
    for (unsigned int arg = declare->num_input; arg < num_args; ++arg) {
        auto* wire_ptr = pin_specs.get(arg);
        if (wire_ptr == nullptr) continue;
        wire_ptr->set_drived();
        auto* wrapped_wire = new OutputWire(wire_ptr);
        wire_map.set(arg, wrapped_wire); output_wires.push_back(wrapped_wire);
    }
    for (unsigned int arg = 0; arg < num_args; arg++) {
        if (wire_map.get(arg) == nullptr) cerr << "| WARNING: Arg (" + to_string(arg) + ") not found in wiremap of cell " << name  << endl;
    }
}

void Cell::push_jobs(queue<Job*>& job_queue) {
    init();
    for (int i = 0; i < schedule_size; ++i) {
        vector<JobHandle*> job_handles; job_handles.reserve(num_args);
        for (int arg = 0; arg < num_args; ++arg) {
            auto* wrapped_wire = wire_map.get(arg);
            if (wrapped_wire == nullptr) job_handles.push_back(new JobHandle());
            else job_handles.push_back(wrapped_wire->get_job_handle(i));
        }
        auto* job = new Job(module_spec, sdf_spec, num_args, job_handles);
        job_queue.emplace(job);
    }
}


void Cell::finish() {
    for (auto& wrapped_wire : output_wires) wrapped_wire->finish();

    hipFree(sdf_spec);
    hipFree(host_sdf_spec.edge_type);
    hipFree(host_sdf_spec.input_index); hipFree(host_sdf_spec.output_index);
    hipFree(host_sdf_spec.rising_delay); hipFree(host_sdf_spec.falling_delay);
    for (auto& wrapped_wire : input_wires) wrapped_wire->free();
    for (auto& wrapped_wire : output_wires) wrapped_wire->free();
}

void Cell::init() {
    set_paths();
    schedule_size = Cell::build_bucket_index_schedule(
        input_wires,
        (INITIAL_CAPACITY * N_STIMULI_PARALLEL) / input_wires.size() - 1
    );
    for (auto& wrapped_wire : output_wires) wrapped_wire->set_schedule_size(schedule_size);
}

unsigned int Cell::build_bucket_index_schedule(vector<InputWire*>& wires, unsigned int size) {
    unsigned int num_finished = 0, num_inputs = wires.size();

    vector<unsigned int> starting_indices; starting_indices.resize(num_inputs);
    vector<bool> finished; finished.resize(num_inputs);
    for (int i_wire = 0; i_wire < num_inputs; i_wire++) {
        const auto& wire = wires[i_wire];
        if (size >= wire->wire->bucket.size()) {
            finished[i_wire] = true;
            num_finished++;
        }
    }

    while (num_finished < num_inputs) {
//        Find min_end_timestamp
        Timestamp min_end_timestamp = LONG_LONG_MAX;
        for(int i_wire = 0; i_wire < num_inputs; i_wire++) {
            auto& wire = wires[i_wire];
            const auto& bucket = wire->wire->bucket;
            unsigned int end_index = starting_indices[i_wire] + size - 1;
            if (end_index >= bucket.size()) continue;
            const auto& end_timestamp = bucket[end_index].timestamp;
            if (end_timestamp < min_end_timestamp) min_end_timestamp = end_timestamp;
        }

        for (int i_wire = 0; i_wire < num_inputs; i_wire++) {
            auto& wire = wires[i_wire];
            const auto& bucket = wire->wire->bucket;
            const auto& bucket_size = bucket.size();
//            If already finished, push_back the last index of bucket
            if (not wire->bucket_index_schedule.empty() and wire->bucket_index_schedule.back() == bucket_size) {
                wire->push_back_schedule_index(bucket_size);
            } else {
//                FIXME will fail if start_index = 0 and timestamp[0] > min_end_timestamp
                auto start_index = bucket[starting_indices[i_wire]].timestamp > min_end_timestamp ? starting_indices[i_wire] - 1 : starting_indices[i_wire];
                auto end_index = find_end_index(bucket, start_index, min_end_timestamp, size);
                auto next_start_index = end_index + 1;
                wire->push_back_schedule_index(next_start_index);
                if (next_start_index + size >= bucket.size() and not finished[i_wire]) {
                    finished[i_wire] = true;
                    num_finished++;
                }
                starting_indices[i_wire] = end_index + 1;
            }
        }
    }
    for (auto& wire : wires) {
        wire->push_back_schedule_index(wire->wire->bucket.size());
    }
    return wires.front()->bucket_index_schedule.size() - 1;
}

unsigned int Cell::find_end_index(const PinnedMemoryVector<Transition>& bucket, unsigned int start_index, const Timestamp& t, unsigned int capacity) {
//    Binary Search for end_index <= t
    unsigned int low = start_index, high = min(start_index + capacity, (unsigned int) bucket.size()) - 1;
    if (bucket[high].timestamp <= t) return high;
    while (low < high) {
        unsigned mid = (low + high) / 2;
        if (mid == low) break;
        if (bucket[mid].timestamp < t) low = mid;
        else if (bucket[mid].timestamp > t) high = mid;
        else return mid;
    }
    return low;
}
