#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <utility>
#include "cell.h"
#include "utils.h"

using namespace std;


Cell::Cell(
    const ModuleSpec* module_spec,
    const StdCellDeclare* declare,
    const WireMap<Wire>& pin_specs,
    Wire* supply1_wire, Wire* supply0_wire,
    string name
) : module_spec(module_spec), name(std::move(name)), num_args(declare->num_args)
{
    build_wire_map(declare, pin_specs, supply1_wire, supply0_wire);
}

Cell::~Cell() {
    for (auto& cell_wire : cell_wires) delete cell_wire->wire;
}

void Cell::set_paths(const vector<SDFPath>& ps) {
    vector<char> edge_types;
    vector<unsigned int> input_indices, output_indices;
    vector<int> rising_delays, falling_delays;

    for (const auto& path : ps) {
        input_indices.push_back(path.in);
        output_indices.push_back(path.out);
        edge_types.push_back(path.edge_type);
        rising_delays.push_back(path.rising_delay);
        falling_delays.push_back(path.falling_delay);
    }

    SDFSpec host_sdf_spec{};
    auto num_rows = ps.size();
    host_sdf_spec.num_rows = num_rows;
    hipMalloc((void**) &host_sdf_spec.edge_type, sizeof(char) * num_rows);
    hipMalloc((void**) &host_sdf_spec.input_index, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.output_index, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.rising_delay, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.falling_delay, sizeof(int) * num_rows);
    hipMemcpy(host_sdf_spec.edge_type, edge_types.data(), sizeof(char) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.input_index, input_indices.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.output_index, output_indices.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.rising_delay, rising_delays.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.falling_delay, falling_delays.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);

    hipMalloc((void**) &sdf_spec, sizeof(SDFSpec));
    hipMemcpy(sdf_spec, &host_sdf_spec, sizeof(SDFSpec), hipMemcpyHostToDevice);
}

void Cell::build_wire_map(
    const StdCellDeclare* declare,
    const WireMap<Wire>& pin_specs,
    Wire *supply1_wire, Wire *supply0_wire
) {
    if (declare->num_args > MAX_NUM_MODULE_ARGS) {
        throw runtime_error("Too many module args (" + to_string(declare->num_args) + ")\n");
    }
    for (const auto& arg: declare->buckets[STD_CELL_INPUT]) {
        auto* wire_ptr = pin_specs.get(arg);
        if (wire_ptr == nullptr) continue;
        auto* scheduled_wire = new ScheduledWire(wire_ptr);
        wire_map.set(arg, scheduled_wire); input_wires.push_back(scheduled_wire);
    }
    for (const auto& arg: declare->buckets[STD_CELL_SUPPLY1]) wire_map.set(arg, new IndexedWire(supply1_wire));
    for (const auto& arg: declare->buckets[STD_CELL_SUPPLY0]) wire_map.set(arg,  new IndexedWire(supply0_wire));

    for (const auto& arg: declare->buckets[STD_CELL_OUTPUT]) {
        auto* wire_ptr = pin_specs.get(arg);
        if (wire_ptr == nullptr) continue;
        auto* indexed_wire = new IndexedWire(wire_ptr);
        wire_map.set(arg, indexed_wire); output_wires.push_back(indexed_wire);
    }
    for (const auto& arg: declare->buckets[STD_CELL_WIRE]) {
        auto* indexed_wire = new IndexedWire(new Wire());
        wire_map.set(arg, indexed_wire); cell_wires.push_back(indexed_wire);
    }
}


void Cell::init() {
    build_scheduled_buckets(input_wires, starting_indices);
}

void Cell::build_scheduled_buckets(vector<ScheduledWire*>& wires, vector<unsigned int>& starting_indices) {
    // initialize indices, num_finished
    unsigned int num_finished = 0; const auto& num_wires = wires.size();

    unsigned int sum_size = 0;
    for (const auto& wire : wires) {
        const auto& size = wire->wire->bucket.size();
        sum_size += size;
        if (size <= 1) num_finished++;
    }
    for (auto& wire : wires) {
        const auto& transitions = wire->wire->bucket.transitions;
        wire->scheduled_bucket.reserve(sum_size);
        wire->scheduled_bucket.push_back(transitions.front());
    }
    starting_indices.reserve(sum_size);
    vector<unsigned int> indices; indices.resize(num_wires);

    // merge sort
    while (num_finished < num_wires) {
        // find min timestamp and corresponding wire index
        Timestamp min_t = LONG_LONG_MAX;
        for (unsigned int i = 0; i < num_wires; i++) {
            if (indices[i] + 1 >= wires[i]->size()) continue;
            const auto& t = wires[i]->wire->bucket[indices[i] + 1].timestamp;
            if (t < min_t) min_t = t;
        }
        assert(min_t != LONG_LONG_MAX);
        starting_indices.push_back(wires.front()->scheduled_bucket.size());

        vector<unsigned int> advancing; advancing.reserve(num_wires);
        for (unsigned int i = 0; i < num_wires; i++) {
            const auto& b = wires[i]->wire->bucket;
            auto& index = indices[i];
            if (b[index + 1].timestamp == min_t) {
                advancing.push_back(i);
                index += 1;
                if (index + 1 >= b.size()) num_finished++;
            }
        }
        for(const auto& advancing_wire_index : advancing) {
            const auto& advancing_inner_bucket = wires[advancing_wire_index]->wire->bucket;
            auto edge_type = get_edge_type(
                advancing_inner_bucket[indices[advancing_wire_index] - 1].value,
                advancing_inner_bucket[indices[advancing_wire_index]].value
            );
            DelayInfo d{advancing_wire_index, edge_type};
            for (unsigned int i = 0; i < num_wires; i++) {
                auto& wire = wires[i];
                wire->scheduled_bucket.emplace_back(min_t, wire->wire->bucket[indices[i]].value, d);
            }
        }
    }
    starting_indices.push_back(wires.front()->scheduled_bucket.size());
}

void Cell::prepare_resource(int session_id, ResourceBuffer& resource_buffer)  {
    resource_buffer.module_specs.push_back(module_spec);
    resource_buffer.sdf_specs.push_back(sdf_spec);
    resource_buffer.data_schedule_offsets.push_back(resource_buffer.data_schedule.size());
    resource_buffer.capacities.push_back(capacity);

    unsigned int progress = 0;
    for (auto& indexed_wire : input_wires) progress = indexed_wire->load(session_id, starting_indices, progress_index);
    for (auto& indexed_wire : output_wires) indexed_wire->load(session_id);
    for (auto& indexed_wire : cell_wires) indexed_wire->load(session_id);
    progress_index = progress;

    for (unsigned int arg = 0; arg < num_args; ++arg) {
        const auto& indexed_wire = wire_map.get(arg);
        if (indexed_wire->first_free_data_ptr_index - 1 >= indexed_wire->data_ptrs.size())
            throw runtime_error("Invalid access to indexed_wire's data_ptrs");
        resource_buffer.data_schedule.push_back(indexed_wire->data_ptrs[indexed_wire->first_free_data_ptr_index - 1]);
    }
}

void Cell::dump_result() {
    for (const auto& indexed_wire : output_wires) indexed_wire->store_to_bucket();
    if (finished()) {
        for (auto& indexed_wire : input_wires) indexed_wire->free();
        for (auto& indexed_wire : output_wires) indexed_wire->free();
        for (auto& indexed_wire : cell_wires) indexed_wire->free();
    }
}

bool Cell::finished() const {
    return progress_index >= starting_indices.size() - 1;
}

