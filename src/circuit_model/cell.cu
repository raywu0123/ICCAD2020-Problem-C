#include <iostream>

#include "cell.h"

using namespace std;


Cell::Cell(
    const ModuleSpec* module_spec,
    const vector<SubmoduleSpec>* submodule_specs,
    const StdCellDeclare* declare,
    const vector<PinSpec> &pin_specs,
    Wire* supply1_wire, Wire* supply0_wire,
    const vector<Wire*>& alloc_wires_param, const vector<Wire*>& free_wires_param
) : module_spec(module_spec)
{
    build_wire_map(declare, pin_specs, supply1_wire, supply0_wire);
    create_wire_schedule(submodule_specs);
    for (const auto& idx : declare->buckets[STD_CELL_INPUT]) {
        input_wires.emplace_back(wire_map[idx]);
    }
    for (const auto& idx : declare->buckets[STD_CELL_OUTPUT]) {
        output_wires.push_back(wire_map[idx]);
    }
}

void Cell::set_paths(const vector<SDFPath>& ps) {
    vector<char> edge_types;
    vector<unsigned int> input_indices, output_indices;
    vector<int> rising_delays, falling_delays;

    for (const auto& path : ps) {
        input_indices.push_back(path.in);
        output_indices.push_back(path.out);
        edge_types.push_back(path.edge_type);
        rising_delays.push_back(path.rising_delay);
        falling_delays.push_back(path.falling_delay);
    }

    SDFSpec host_sdf_spec{};
    auto num_rows = ps.size();
    host_sdf_spec.num_rows = num_rows;
    hipMalloc((void**) &host_sdf_spec.edge_type, sizeof(char) * num_rows);
    hipMalloc((void**) &host_sdf_spec.input_index, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.output_index, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.rising_delay, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.falling_delay, sizeof(int) * num_rows);
    hipMemcpy(host_sdf_spec.edge_type, edge_types.data(), sizeof(char) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.input_index, input_indices.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.output_index, output_indices.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.rising_delay, rising_delays.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.falling_delay, falling_delays.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);

    hipMalloc((void**) &sdf_spec, sizeof(SDFSpec));
    hipMemcpy(sdf_spec, &host_sdf_spec, sizeof(SDFSpec), hipMemcpyHostToDevice);
}

Cell::~Cell() {
    for (auto& wire_ptr: cell_wires) {
        delete wire_ptr;
    }
}

void Cell::build_wire_map(
    const StdCellDeclare* declare,
    const vector<PinSpec> &pin_specs,
    Wire *supply1_wire, Wire *supply0_wire
)
{
    if (not wire_map.empty()) throw runtime_error("wire_map not empty.");

    for (const auto& pin_spec: pin_specs) wire_map[pin_spec.index] = pin_spec.wire;
    for (const auto& arg: declare->buckets[STD_CELL_SUPPLY1]) wire_map[arg] = supply1_wire;
    for (const auto& arg: declare->buckets[STD_CELL_SUPPLY0]) wire_map[arg] = supply0_wire;
}


void Cell::create_wire_schedule(
    const vector<SubmoduleSpec>* submodule_specs
)  {
    for(const auto& submodule_spec: *submodule_specs) {
        for (const auto& arg: submodule_spec.args) {
            const auto& it = wire_map.find(arg);
            if (it != wire_map.end()) {
                wire_schedule.emplace_back(it->second);
            } else {
                // create cell wire
                auto* wire_ptr = new Wire();
                wire_schedule.emplace_back(wire_ptr);
                add_cell_wire(wire_ptr);
            }
        }
    }
}

void Cell::add_cell_wire(Wire *wire_ptr) {
    cell_wires.push_back(wire_ptr);
}

void Cell::build_bucket_index_schedule() {
//    TODO
}

bool Cell::prepare_resource(ResourceBuffer& resource_buffer)  {
    resource_buffer.module_specs.push_back(module_spec);
    resource_buffer.sdf_specs.push_back(sdf_spec);
    resource_buffer.data_schedule_offsets.push_back(resource_buffer.data_schedule_offsets.size());

//    allocate data memory
    for (const auto& wire : wire_schedule) {
        auto* data_ptr = wire->alloc();
        resource_buffer.data_schedule.push_back(data_ptr);
        resource_buffer.capacities.push_back(wire->capacity);
    }

    bool all_finished = true;
    for (auto& indexed_wire : input_wires) {
        all_finished &= indexed_wire.load_from_bucket();
    }
    return all_finished;
}

void Cell::finalize() {
    for (const auto& wire : wire_schedule) {
        wire->free();
    }
}
