#include "hip/hip_runtime.h"
#include <iostream>
#include <utility>
#include "cell.h"

using namespace std;


Cell::Cell(
    const ModuleSpec* module_spec,
    const vector<SubmoduleSpec>* submodule_specs,
    const StdCellDeclare* declare,
    const vector<PinSpec> &pin_specs,
    Wire* supply1_wire, Wire* supply0_wire,
    string name
) : module_spec(module_spec), name(std::move(name))
{
    hipMalloc((void**) &overflow_ptr, sizeof(bool));

    build_wire_map(declare, pin_specs, supply1_wire, supply0_wire);
    create_wire_schedule(submodule_specs);
}

Cell::~Cell() {
    for (auto& cell_wire : cell_wires) delete cell_wire->wire;
    for (auto& it : wire_map) delete it.second;
}

void Cell::set_paths(const vector<SDFPath>& ps) {
    vector<char> edge_types;
    vector<unsigned int> input_indices, output_indices;
    vector<int> rising_delays, falling_delays;

    for (const auto& path : ps) {
        input_indices.push_back(path.in);
        output_indices.push_back(path.out);
        edge_types.push_back(path.edge_type);
        rising_delays.push_back(path.rising_delay);
        falling_delays.push_back(path.falling_delay);
    }

    SDFSpec host_sdf_spec{};
    auto num_rows = ps.size();
    host_sdf_spec.num_rows = num_rows;
    hipMalloc((void**) &host_sdf_spec.edge_type, sizeof(char) * num_rows);
    hipMalloc((void**) &host_sdf_spec.input_index, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.output_index, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.rising_delay, sizeof(int) * num_rows);
    hipMalloc((void**) &host_sdf_spec.falling_delay, sizeof(int) * num_rows);
    hipMemcpy(host_sdf_spec.edge_type, edge_types.data(), sizeof(char) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.input_index, input_indices.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.output_index, output_indices.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.rising_delay, rising_delays.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(host_sdf_spec.falling_delay, falling_delays.data(), sizeof(int) * num_rows, hipMemcpyHostToDevice);

    hipMalloc((void**) &sdf_spec, sizeof(SDFSpec));
    hipMemcpy(sdf_spec, &host_sdf_spec, sizeof(SDFSpec), hipMemcpyHostToDevice);
}

void Cell::build_wire_map(
    const StdCellDeclare* declare,
    const vector<PinSpec> &pin_specs,
    Wire *supply1_wire, Wire *supply0_wire
)
{
    if (not wire_map.empty()) throw runtime_error("wire_map not empty.");
    unordered_map<unsigned int, Wire*> index_to_wire_ptr;
    for (const auto& pin_spec: pin_specs) index_to_wire_ptr.emplace(pin_spec.index, pin_spec.wire);

    for (const auto& arg: declare->buckets[STD_CELL_INPUT]) {
        bool specified = index_to_wire_ptr.find(arg) != index_to_wire_ptr.end();
        if (not specified) continue;
        auto* scheduled_wire = new ScheduledWire(index_to_wire_ptr[arg]);
        wire_map.emplace(arg, scheduled_wire); input_wires.push_back(scheduled_wire);
    }
    for (const auto& arg: declare->buckets[STD_CELL_SUPPLY1]) wire_map.emplace(arg, new IndexedWire(supply1_wire));
    for (const auto& arg: declare->buckets[STD_CELL_SUPPLY0]) wire_map.emplace(arg, new IndexedWire(supply0_wire));

    for (const auto& arg: declare->buckets[STD_CELL_OUTPUT]) {
        bool specified = index_to_wire_ptr.find(arg) != index_to_wire_ptr.end();
        if (not specified) continue;
        auto* indexed_wire = new IndexedWire(index_to_wire_ptr[arg]);
        wire_map.emplace(arg, indexed_wire); output_wires.push_back(indexed_wire);
    }
    for (const auto& arg: declare->buckets[STD_CELL_WIRE]) {
        auto* indexed_wire = new IndexedWire(new Wire());
        wire_map.emplace(arg, indexed_wire); cell_wires.push_back(indexed_wire);
    }
}

void Cell::create_wire_schedule(const vector<SubmoduleSpec>* submodule_specs)  {
    for(const auto& submodule_spec: *submodule_specs) {
        for (const auto& arg: submodule_spec.args) {
            const auto& it = wire_map.find(arg);
            if (it != wire_map.end()) wire_schedule.emplace_back(it->second);
            else throw runtime_error("Wire not found in wire_map.");
        }
    }
}

void Cell::build_bucket_index_schedule(vector<ScheduledWire*>& wires, unsigned int capacity) {
    unsigned int num_finished = 0;
    unsigned int num_inputs = wires.size();

    vector<unsigned int> starting_indices; starting_indices.resize(num_inputs);
    vector<bool> finished; finished.resize(num_inputs);
    for (int i_wire = 0; i_wire < num_inputs; i_wire++) {
        const auto& wire = wires[i_wire];
        if (capacity >= wire->wire->bucket.size()) {
            finished[i_wire] = true;
            num_finished++;
        }
    }

    while (num_finished < num_inputs) {
//        Find min_end_timestamp
        Timestamp min_end_timestamp = LONG_LONG_MAX;
        for(int i_wire = 0; i_wire < num_inputs; i_wire++) {
            auto& wire = wires[i_wire];
            const auto& bucket = wire->wire->bucket;
            unsigned int end_index = starting_indices[i_wire] + capacity - 1;
            if (end_index >= bucket.size()) continue;
            const auto& end_timestamp = bucket.transitions[end_index].timestamp;
            if (end_timestamp < min_end_timestamp) min_end_timestamp = end_timestamp;
        }

        for (int i_wire = 0; i_wire < num_inputs; i_wire++) {
            auto& wire = wires[i_wire];
            const auto& bucket = wire->wire->bucket;
//            If already finished, push_back the last index of bucket
            if (
                not wire->bucket_index_schedule.empty()
                and wire->bucket_index_schedule.back() == bucket.size()
            ) wire->push_back_schedule_index(bucket.size());
            else {
//                FIXME will fail if start_index = 0 and timestamp[0] > min_end_timestamp
                auto start_index = bucket.transitions[starting_indices[i_wire]].timestamp > min_end_timestamp ? starting_indices[i_wire] - 1 : starting_indices[i_wire];
                auto end_index = find_end_index(bucket, start_index, min_end_timestamp, capacity);
                auto next_start_index = end_index + 1;
                wire->push_back_schedule_index(next_start_index);
                if (next_start_index + capacity >= bucket.size() and not finished[i_wire]) {
                    finished[i_wire] = true;
                    num_finished++;
                }
                starting_indices[i_wire] = end_index + 1;
            }
        }
    }
    for (auto& wire : wires) {
        wire->push_back_schedule_index(wire->wire->bucket.size());
    }
}

unsigned int Cell::find_end_index(const Bucket& bucket, unsigned int start_index, Timestamp t, unsigned int capacity) {
//    Binary Search for end_index <= t
    unsigned int low = start_index;
    unsigned int high = min(start_index + capacity, bucket.size()) - 1;
    if (bucket.transitions[high].timestamp <= t) return high;
    while (low < high) {
        unsigned mid = (low + high) / 2;
        if (mid == low) break;
        if (bucket.transitions[mid].timestamp < t) low = mid;
        else if (bucket.transitions[mid].timestamp > t) high = mid;
        else return mid;
    }
    return low;
}

void Cell::init() {
    Cell::build_bucket_index_schedule(input_wires, INITIAL_CAPACITY - 1); // leave one for delay calculation
}

void Cell::prepare_resource(int session_id, ResourceBuffer& resource_buffer)  {
    hipMemset(overflow_ptr, 0, sizeof(bool)); // reset overflow value
    resource_buffer.overflows.push_back(overflow_ptr);

    resource_buffer.module_specs.push_back(module_spec);
    resource_buffer.sdf_specs.push_back(sdf_spec);
    resource_buffer.data_schedule_offsets.push_back(resource_buffer.data_schedule.size());

    for (auto& indexed_wire : input_wires) indexed_wire->load(session_id);
    for (auto& indexed_wire : output_wires) indexed_wire->load(session_id);
    for (auto& indexed_wire : cell_wires) indexed_wire->load(session_id);

    for (auto& indexed_wire : wire_schedule) {
        if (indexed_wire->first_free_data_ptr_index - 1 >= indexed_wire->data_ptrs.size()) throw runtime_error("invalid access to indexed_wire's data_ptrs");
        resource_buffer.data_schedule.emplace_back(indexed_wire->data_ptrs[indexed_wire->first_free_data_ptr_index - 1], indexed_wire->capacity);
    }
}

void Cell::dump_result() {
    for (const auto& indexed_wire : output_wires) indexed_wire->store_to_bucket();
    if (finished()) {
        for (auto& indexed_wire : input_wires) indexed_wire->free();
        for (auto& indexed_wire : output_wires) indexed_wire->free();
        for (auto& indexed_wire : cell_wires) indexed_wire->free();
    }
}

void Cell::handle_overflow() {
    for (auto& indexed_wire : input_wires) indexed_wire->handle_overflow();
    for (auto& indexed_wire : cell_wires) indexed_wire->handle_overflow();
    for (auto& indexed_wire : output_wires) indexed_wire->handle_overflow();
}

bool Cell::overflow() const {
    bool host_overflow_value;
    hipMemcpy(&host_overflow_value, overflow_ptr, sizeof(bool), hipMemcpyDeviceToHost);
    return host_overflow_value;
}

bool Cell::finished() const {
    bool finished = true;
    for (auto& indexed_wire : input_wires) finished &= indexed_wire->finished();
    return finished;
}
