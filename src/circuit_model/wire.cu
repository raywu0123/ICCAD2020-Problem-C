#include <cassert>

#include "wire.h"
#include "utils.h"

using namespace std;

Wire::Wire(const WireInfo& wire_info) {
    wire_infos.push_back(wire_info);
}

// memory pattern
//      0                   1   2   3  ...    C
// 0   previous_transition t00 t01 t02 ... t0(c-1)
// 1   t0(c-1)             t10 t11 t12
// 2   t1(c-1)
// .
// .
// N-1


void Wire::assign(const Wire& other_wire) {
    wire_infos.insert(wire_infos.end(), other_wire.wire_infos.begin(), other_wire.wire_infos.end());
}

void Wire::load_from_bucket(
    Transition* ptr, unsigned int start_bucket_index, unsigned int end_bucket_index
) {
    auto status = hipMemcpyAsync(
        ptr,
        bucket.transitions.data() + start_bucket_index,
        sizeof(Transition) * (end_bucket_index - start_bucket_index),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) throw runtime_error(hipGetErrorString(status));
}

void Wire::store_to_bucket(const vector<Data>& data_list, unsigned int num_ptrs) {
    assert(num_ptrs <= data_list.size());
    for (unsigned int i = 0; i < num_ptrs; i++) bucket.push_back(data_list[i]);
}

void Wire::set_drived() {
    bucket.transitions[0].value = Values::X;
}

ConstantWire::ConstantWire(Values value): value(value) {
    bucket.transitions.clear();
    bucket.transitions.emplace_back(0, value);
}
