#include <cassert>

#include "containers.h"

using namespace std;

void BatchResource::init(hipStream_t) {
    hipMalloc((void**) &overflows, sizeof(bool*) * N_CELL_PARALLEL);
    hipMalloc((void**) &capacities, sizeof(CAPACITY_TYPE) * N_CELL_PARALLEL);
    hipMalloc((void**) &module_specs, sizeof(ModuleSpec*) * N_CELL_PARALLEL);
    hipMalloc((void**) &sdf_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &s_timestamp_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &s_delay_info_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &s_value_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &s_length_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);

    hipMalloc((void**) &sdf_num_rows, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &input_data_schedule, sizeof(InputData) * N_CELL_PARALLEL * MAX_NUM_MODULE_INPUT);
    hipMalloc((void**) &output_data_schedule, sizeof(Data) * N_CELL_PARALLEL * MAX_NUM_MODULE_OUTPUT);
}

void BatchResource::set(const ResourceBuffer& resource_buffer, hipStream_t stream) {
    assert(resource_buffer.input_data_schedule.size() <= N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
    assert(resource_buffer.output_data_schedule.size() <= N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
    num_modules = resource_buffer.size;

    auto direction = hipMemcpyHostToDevice;
    hipMemcpyAsync(overflows, resource_buffer.overflows.data(), sizeof(bool*) * num_modules, direction, stream);
    hipMemcpyAsync(capacities, resource_buffer.capacities.data(), sizeof(CAPACITY_TYPE) * num_modules, direction, stream);
    hipMemcpyAsync(module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, direction, stream);
    hipMemcpyAsync(sdf_offsets, resource_buffer.sdf_offsets.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(s_timestamp_offsets, resource_buffer.s_timestamp_offsets.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(s_delay_info_offsets, resource_buffer.s_delay_info_offsets.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(s_value_offsets, resource_buffer.s_value_offsets.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(s_length_offsets, resource_buffer.s_length_offsets.data(), sizeof(unsigned int) * num_modules, direction, stream);

    hipMemcpyAsync(sdf_num_rows, resource_buffer.sdf_num_rows.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(input_data_schedule, resource_buffer.input_data_schedule.data(), sizeof(InputData) * resource_buffer.input_data_schedule.size(), direction, stream);
    hipMemcpyAsync(output_data_schedule, resource_buffer.output_data_schedule.data(), sizeof(Data) * resource_buffer.output_data_schedule.size(), direction, stream);
}

void BatchResource::free() const {
    hipFree(overflows);
    hipFree(capacities);
    hipFree(module_specs);
    hipFree(sdf_offsets);
    hipFree(s_timestamp_offsets); hipFree(s_delay_info_offsets); hipFree(s_value_offsets); hipFree(s_length_offsets);
    hipFree(sdf_num_rows);
    hipFree(input_data_schedule); hipFree(output_data_schedule);
}

ResourceBuffer::ResourceBuffer() {
    overflows.reserve(N_CELL_PARALLEL);
    capacities.reserve(N_CELL_PARALLEL);
    module_specs.reserve(N_CELL_PARALLEL);
    sdf_offsets.reserve(N_CELL_PARALLEL);
    s_timestamp_offsets.reserve(N_CELL_PARALLEL); s_delay_info_offsets.reserve(N_CELL_PARALLEL); s_value_offsets.reserve(N_CELL_PARALLEL); s_length_offsets.reserve(N_CELL_PARALLEL);
    sdf_num_rows.reserve(N_CELL_PARALLEL);
    input_data_schedule.reserve(N_CELL_PARALLEL * MAX_NUM_MODULE_INPUT);
    output_data_schedule.reserve(N_CELL_PARALLEL * MAX_NUM_MODULE_OUTPUT);
}

void ResourceBuffer::finish_module() {
    size++;
    input_data_schedule.resize(size * MAX_NUM_MODULE_INPUT);
    output_data_schedule.resize(size * MAX_NUM_MODULE_OUTPUT);
}

void ResourceBuffer::clear() {
    overflows.clear();
    capacities.clear();
    module_specs.clear();
    sdf_offsets.clear(); s_timestamp_offsets.clear(); s_delay_info_offsets.clear(); s_value_offsets.clear(); s_length_offsets.clear();
    sdf_num_rows.clear();
    input_data_schedule.clear();
    output_data_schedule.clear();
    size = 0;
}

