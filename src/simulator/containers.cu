#include "containers.h"


void BatchResource::init(const ResourceBuffer& resource_buffer, const hipStream_t& stream) {
    num_modules = resource_buffer.size;

    hipMalloc((void**) &overflows, sizeof(bool*) * num_modules);
    hipMalloc((void**) &capacities, sizeof(unsigned int) * num_modules);
    hipMalloc((void**) &module_specs, sizeof(ModuleSpec*) * num_modules);
    hipMalloc((void**) &sdf_specs, sizeof(SDFSpec*) * num_modules);
    hipMalloc((void**) &data_schedule, sizeof(Data) * resource_buffer.data_schedule.size());

    auto direction = hipMemcpyHostToDevice;
    hipMemcpyAsync(overflows, resource_buffer.overflows.data(), sizeof(bool*) * num_modules, direction, stream);
    hipMemcpyAsync(capacities, resource_buffer.capacities.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, direction, stream);
    hipMemcpyAsync(sdf_specs, resource_buffer.sdf_specs.data(), sizeof(SDFSpec*) * num_modules, direction, stream);
    hipMemcpyAsync(data_schedule, resource_buffer.data_schedule.data(), sizeof(Data) * resource_buffer.data_schedule.size(), direction, stream);
}

void BatchResource::free() const {
    hipFree(overflows);
    hipFree(capacities);
    hipFree(module_specs);
    hipFree(sdf_specs);
    hipFree(data_schedule);
}

ResourceBuffer::ResourceBuffer() {
    overflows.reserve(N_CELL_PARALLEL);
    capacities.reserve(N_CELL_PARALLEL);
    module_specs.reserve(N_CELL_PARALLEL);
    sdf_specs.reserve(N_CELL_PARALLEL);
    data_schedule.reserve(N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
}

void ResourceBuffer::finish_module() {
    size++;
    data_schedule.resize(size * MAX_NUM_MODULE_ARGS);
}
