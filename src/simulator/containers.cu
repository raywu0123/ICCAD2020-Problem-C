#include <cassert>

#include "containers.h"

using namespace std;

void BatchResource::init(hipStream_t) {
    hipMalloc((void**) &overflows, sizeof(bool*) * N_CELL_PARALLEL);
    hipMalloc((void**) &capacities, sizeof(CAPACITY_TYPE) * N_CELL_PARALLEL);
    hipMalloc((void**) &module_specs, sizeof(ModuleSpec*) * N_CELL_PARALLEL);
    hipMalloc((void**) &sdf_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &sdf_num_rows, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &input_data_schedule, sizeof(InputData) * N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
    hipMalloc((void**) &output_data_schedule, sizeof(Data) * N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
}

void BatchResource::set(const ResourceBuffer& resource_buffer, hipStream_t stream) {
    assert(resource_buffer.input_data_schedule.size() <= N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
    assert(resource_buffer.output_data_schedule.size() <= N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
    num_modules = resource_buffer.size;

    auto direction = hipMemcpyHostToDevice;
    hipMemcpyAsync(overflows, resource_buffer.overflows.data(), sizeof(bool*) * num_modules, direction);
    hipMemcpyAsync(capacities, resource_buffer.capacities.data(), sizeof(CAPACITY_TYPE) * num_modules, direction);
    hipMemcpyAsync(module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, direction);
    hipMemcpyAsync(sdf_offsets, resource_buffer.sdf_offsets.data(), sizeof(unsigned int) * num_modules, direction);
    hipMemcpyAsync(sdf_num_rows, resource_buffer.sdf_num_rows.data(), sizeof(unsigned int) * num_modules, direction);
    hipMemcpyAsync(input_data_schedule, resource_buffer.input_data_schedule.data(), sizeof(InputData) * resource_buffer.input_data_schedule.size(), direction);
    hipMemcpyAsync(output_data_schedule, resource_buffer.output_data_schedule.data(), sizeof(Data) * resource_buffer.output_data_schedule.size(), direction);
}

void BatchResource::free() const {
    hipFree(overflows);
    hipFree(capacities);
    hipFree(module_specs);
    hipFree(sdf_offsets);
    hipFree(sdf_num_rows);
    hipFree(input_data_schedule); hipFree(output_data_schedule);
}

ResourceBuffer::ResourceBuffer() {
    overflows.reserve(N_CELL_PARALLEL);
    capacities.reserve(N_CELL_PARALLEL);
    module_specs.reserve(N_CELL_PARALLEL);
    sdf_offsets.reserve(N_CELL_PARALLEL);
    sdf_num_rows.reserve(N_CELL_PARALLEL);
    input_data_schedule.reserve(N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
    output_data_schedule.reserve(N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
}

void ResourceBuffer::finish_module() {
    size++;
    input_data_schedule.resize(size * MAX_NUM_MODULE_ARGS);
    output_data_schedule.resize(size * MAX_NUM_MODULE_ARGS);
}

void ResourceBuffer::clear() {
    overflows.clear();
    capacities.clear();
    module_specs.clear();
    sdf_offsets.clear();
    sdf_num_rows.clear();
    input_data_schedule.clear();
    output_data_schedule.clear();
    size = 0;
}
