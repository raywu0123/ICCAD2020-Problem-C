#include <cassert>

#include "containers.h"


void BatchResource::init(hipStream_t) {
    hipMalloc((void**) &overflows, sizeof(bool*) * N_CELL_PARALLEL);
    hipMalloc((void**) &capacities, sizeof(CAPACITY_TYPE) * N_CELL_PARALLEL);
    hipMalloc((void**) &module_specs, sizeof(ModuleSpec*) * N_CELL_PARALLEL);
    hipMalloc((void**) &sdf_specs, sizeof(SDFSpec*) * N_CELL_PARALLEL);
    hipMalloc((void**) &data_schedule, sizeof(Data) * N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
}

void BatchResource::set(const ResourceBuffer& resource_buffer, hipStream_t stream) {
    assert(resource_buffer.data_schedule.size() <= N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
    num_modules = resource_buffer.size;

    auto direction = hipMemcpyHostToDevice;
    hipMemcpyAsync(overflows, resource_buffer.overflows.data(), sizeof(bool*) * num_modules, direction);
    hipMemcpyAsync(capacities, resource_buffer.capacities.data(), sizeof(CAPACITY_TYPE) * num_modules, direction);
    hipMemcpyAsync(module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, direction);
    hipMemcpyAsync(sdf_specs, resource_buffer.sdf_specs.data(), sizeof(SDFSpec*) * num_modules, direction);
    hipMemcpyAsync(data_schedule, resource_buffer.data_schedule.data(), sizeof(Data) * resource_buffer.data_schedule.size(), direction);
}

void BatchResource::free() const {
    hipFree(overflows);
    hipFree(capacities);
    hipFree(module_specs);
    hipFree(sdf_specs);
    hipFree(data_schedule);
}

ResourceBuffer::ResourceBuffer() {
    overflows.reserve(N_CELL_PARALLEL);
    capacities.reserve(N_CELL_PARALLEL);
    module_specs.reserve(N_CELL_PARALLEL);
    sdf_specs.reserve(N_CELL_PARALLEL);
    data_schedule.reserve(N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
}

void ResourceBuffer::finish_module() {
    size++;
    data_schedule.resize(size * MAX_NUM_MODULE_ARGS);
}

void ResourceBuffer::clear() {
    overflows.clear();
    capacities.clear();
    module_specs.clear();
    sdf_specs.clear();
    data_schedule.clear();
    size = 0;
}
