#include <cassert>

#include "containers.h"

using namespace std;

void BatchResource::init(hipStream_t) {
    hipMalloc((void**) &output_capacities, sizeof(CAPACITY_TYPE) * N_CELL_PARALLEL);
    hipMalloc((void**) &s_capacities, sizeof(CAPACITY_TYPE) * N_CELL_PARALLEL);
    hipMalloc((void**) &module_specs, sizeof(ModuleSpec*) * N_CELL_PARALLEL);
    hipMalloc((void**) &sdf_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &overflow_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &s_overflow_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &s_slice_info_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &s_delay_info_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &s_length_offsets, sizeof(unsigned int) * N_CELL_PARALLEL);

    hipMalloc((void**) &sdf_num_rows, sizeof(unsigned int) * N_CELL_PARALLEL);
    hipMalloc((void**) &input_data_schedule, sizeof(InputData) * N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
    hipMalloc((void**) &output_data_schedule, sizeof(Data) * N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
}

void BatchResource::set(const ResourceBuffer& resource_buffer, hipStream_t stream) {
    assert(resource_buffer.input_data_schedule.size() <= N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
    assert(resource_buffer.output_data_schedule.size() <= N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
    num_modules = resource_buffer.size;

    auto direction = hipMemcpyHostToDevice;
    hipMemcpyAsync(output_capacities, resource_buffer.output_capacities.data(), sizeof(CAPACITY_TYPE) * num_modules, direction, stream);
    hipMemcpyAsync(s_capacities, resource_buffer.s_capacities.data(), sizeof(CAPACITY_TYPE) * num_modules, direction, stream);
    hipMemcpyAsync(module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, direction, stream);
    hipMemcpyAsync(sdf_offsets, resource_buffer.sdf_offsets.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(overflow_offsets, resource_buffer.overflow_offsets.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(s_overflow_offsets, resource_buffer.s_overflow_offsets.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(s_slice_info_offsets, resource_buffer.s_slice_info_offsets.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(s_delay_info_offsets, resource_buffer.s_delay_info_offsets.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(s_length_offsets, resource_buffer.s_length_offsets.data(), sizeof(unsigned int) * num_modules, direction, stream);

    hipMemcpyAsync(sdf_num_rows, resource_buffer.sdf_num_rows.data(), sizeof(unsigned int) * num_modules, direction, stream);
    hipMemcpyAsync(input_data_schedule, resource_buffer.input_data_schedule.data(), sizeof(InputData) * resource_buffer.input_data_schedule.size(), direction, stream);
    hipMemcpyAsync(output_data_schedule, resource_buffer.output_data_schedule.data(), sizeof(Data) * resource_buffer.output_data_schedule.size(), direction, stream);
}

void BatchResource::free() const {
    hipFree(output_capacities); hipFree(s_capacities);
    hipFree(module_specs);
    hipFree(sdf_offsets);
    hipFree(overflow_offsets); hipFree(s_overflow_offsets);
    hipFree(s_slice_info_offsets); hipFree(s_delay_info_offsets); hipFree(s_length_offsets);
    hipFree(sdf_num_rows);
    hipFree(input_data_schedule); hipFree(output_data_schedule);
}

ResourceBuffer::ResourceBuffer() {
    output_capacities.reserve(N_CELL_PARALLEL); s_capacities.reserve(N_CELL_PARALLEL);
    module_specs.reserve(N_CELL_PARALLEL);
    sdf_offsets.reserve(N_CELL_PARALLEL);
    overflow_offsets.reserve(N_CELL_PARALLEL); s_overflow_offsets.reserve(N_CELL_PARALLEL);
    s_slice_info_offsets.reserve(N_CELL_PARALLEL); s_delay_info_offsets.reserve(N_CELL_PARALLEL); s_length_offsets.reserve(N_CELL_PARALLEL);
    sdf_num_rows.reserve(N_CELL_PARALLEL);
    input_data_schedule.reserve(N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
    output_data_schedule.reserve(N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
}

void ResourceBuffer::finish_module() {
    size++;
    input_data_schedule.resize(size * MAX_NUM_MODULE_ARGS);
    output_data_schedule.resize(size * MAX_NUM_MODULE_ARGS);
}

void ResourceBuffer::clear() {
    output_capacities.clear(); s_capacities.clear();
    module_specs.clear();
    overflow_offsets.clear(); s_overflow_offsets.clear();
    sdf_offsets.clear(); s_slice_info_offsets.clear(); s_delay_info_offsets.clear(); s_length_offsets.clear();
    sdf_num_rows.clear();
    input_data_schedule.clear();
    output_data_schedule.clear();
    size = 0;
}

