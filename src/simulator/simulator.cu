#include "hip/hip_runtime.h"
#include <stack>
#include <cassert>

#include "simulator/simulator.h"
#include "simulator/collision_utils.h"
#include "include/progress_bar.h"

using namespace std;


__device__ __host__ void resolve_collisions_for_batch_stimuli(
    Transition** data,
    const unsigned int* lengths,
    const unsigned int capacity,
    const unsigned int num_inputs, const unsigned int num_outputs
) {
//    TODO parallelize
    unsigned int stimuli_lengths[N_STIMULI_PARALLEL];
    for (int i_output = 0; i_output < num_outputs; i_output++) {
        for(int i_stimuli = 0; i_stimuli < N_STIMULI_PARALLEL; i_stimuli++) {
            stimuli_lengths[i_stimuli] = lengths[num_outputs * i_stimuli + i_output];
            assert(stimuli_lengths[i_stimuli] <= capacity);
        }
        resolve_collisions_for_batch_waveform(
            data[num_inputs + i_output], capacity,
            stimuli_lengths, N_STIMULI_PARALLEL
        );
    }
}

__device__ void init_delay_info(Transition** data, unsigned num_input) {
    for (unsigned int i = 0; i < num_input; i++) {
        for (unsigned int j = 1; j < INITIAL_CAPACITY; j++) {
             if (data[i][j].value == 0) break;
             data[i][j].delay_info.arg = i;
             data[i][j].delay_info.edge_type = get_edge_type(data[i][j - 1].value, data[i][j].value);
        }
    }
}

__device__ void simulate_module(
    const ModuleSpec* const module_spec,
    const SDFSpec* const sdf_spec,
    Transition** const data,
    const unsigned int capacity,
    bool* overflow_ptr
) {
    unsigned stimuli_idx = threadIdx.x;
    Transition* data_ptrs_for_each_stimuli[MAX_NUM_MODULE_ARGS];
    for (unsigned int i = 0; i < module_spec->num_module_args; i++) {
        data_ptrs_for_each_stimuli[i] = data[i] + stimuli_idx * capacity;
    }
    init_delay_info(data_ptrs_for_each_stimuli, module_spec->num_module_input);

    unsigned int offset = 0;
    for (int i = 0; i < module_spec->schedule_size; i++) {
        const unsigned int num_gate_args = module_spec->num_inputs[i] + module_spec->num_outputs[i];
        assert(num_gate_args <= MAX_NUM_GATE_ARGS);
        Transition* data_schedule_for_gate[MAX_NUM_GATE_ARGS] = { nullptr };
        for (int j = 0; j < num_gate_args; ++j) {
            const auto& arg = module_spec->gate_specs[offset + j];
            data_schedule_for_gate[j] = data_ptrs_for_each_stimuli[arg];
        }
        module_spec->gate_schedule[i](
            data_schedule_for_gate,
            capacity,
            module_spec->tables[i], module_spec->table_row_num[i],
            module_spec->num_inputs[i], module_spec->num_outputs[i],
            overflow_ptr
        );
        offset += num_gate_args;
    }
    assert(module_spec->num_module_output <= MAX_NUM_MODULE_OUTPUT);
    __shared__ unsigned int lengths[N_STIMULI_PARALLEL * MAX_NUM_MODULE_OUTPUT];
    compute_delay(
        data_ptrs_for_each_stimuli, capacity,
        module_spec->num_module_output, module_spec->num_module_input,
        sdf_spec, lengths + stimuli_idx * module_spec->num_module_output
    );

    __syncthreads();
    if (threadIdx.x == 0) {
        resolve_collisions_for_batch_stimuli(
            data, lengths, capacity,
            module_spec->num_module_input, module_spec->num_module_output
        );
    }
}

__global__ void simulate_batch(BatchResource batch_resource) {
    if (blockIdx.x < batch_resource.num_modules) {
        const auto& offset = batch_resource.data_schedule_offsets[blockIdx.x];
        const auto& module_spec = batch_resource.module_specs[blockIdx.x];
        const auto& sdf_spec = batch_resource.sdf_specs[blockIdx.x];
        auto* module_data = &batch_resource.data_schedule[offset];
        const auto& capacity = batch_resource.capacities[blockIdx.x];
        auto* overflow_ptr = batch_resource.overflows[blockIdx.x];
        simulate_module(module_spec, sdf_spec, module_data, capacity, overflow_ptr);
    }
}

void Simulator::run() {
    cout << "| Status: Running Simulation... " << endl;

    unsigned int num_layers = circuit.cell_schedule.size();
    cout << "| Total " << num_layers << " layers" << endl;

    ProgressBar progress_bar(num_layers);
    for (unsigned int i_layer = 0; i_layer < num_layers; i_layer++) {
        const auto& schedule_layer = circuit.cell_schedule[i_layer];
        for (auto* cell : schedule_layer) cell->init();
        stack<Cell*, std::vector<Cell*>> job_queue(schedule_layer);
        int session_id = 0;

        while (not job_queue.empty()) {
            unordered_set<Cell*> processing_cells;
            ResourceBuffer resource_buffer;
            for (int i = 0; i < N_CELL_PARALLEL; i++) {
                if (job_queue.empty()) break;
                auto* cell = job_queue.top(); processing_cells.insert(cell);
                cell->prepare_resource(session_id, resource_buffer);
                if (cell->finished()) job_queue.pop();
            }
            BatchResource batch_data{}; batch_data.init(resource_buffer);
            simulate_batch<<<N_CELL_PARALLEL, N_STIMULI_PARALLEL>>>(batch_data);
            hipDeviceSynchronize();

            for (auto* cell : processing_cells) {
                if (cell->overflow()) {
                    if (cell->finished()) job_queue.push(cell);
                    cell->handle_overflow();
                }
                else cell->dump_result();
            }
            batch_data.free();
            session_id++;
        }
        progress_bar.Progressed(i_layer + 1);
    }
    cout << endl;
}
