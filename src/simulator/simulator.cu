#include "hip/hip_runtime.h"
#include <stack>
#include <cassert>

#include "simulator/simulator.h"
#include "simulator/collision_utils.h"
#include "include/progress_bar.h"

using namespace std;


__device__ __host__ void resolve_collisions_for_batch_stimuli(
    Transition** data,
    const unsigned int* lengths, unsigned int capacity,
    const unsigned int num_inputs, const unsigned int num_outputs
) {
//    TODO parallelize
    unsigned int stimuli_lengths[N_STIMULI_PARALLEL];
    for (int i_output = 0; i_output < num_outputs; i_output++) {
        for(int i_stimuli = 0; i_stimuli < N_STIMULI_PARALLEL; i_stimuli++) {
            stimuli_lengths[i_stimuli] = lengths[num_outputs * i_stimuli + i_output];
            assert(stimuli_lengths[i_stimuli] <= capacity);
        }
        resolve_collisions_for_batch_waveform(
            data[num_inputs + i_output],
            stimuli_lengths, capacity,
            N_STIMULI_PARALLEL
        );
    }
}


__device__ __host__ bool OOB(unsigned int index, Transition** const data, unsigned int i) {
    return index >= N_STIMULI_PARALLEL * INITIAL_CAPACITY or data[i][index].value == 0;
}

__device__ __host__ void prepare_stimuli_head(
    Timestamp* s_timestamps, char* s_values,
    Transition** data,
    const unsigned int num_wires, const unsigned int* progress_updates
) {
    s_timestamps[0] = data[0][progress_updates[0]].timestamp;
    for (int i = 0; i < num_wires; ++i) {
        s_values[i] = data[i][progress_updates[i]].value;
    }
}

__device__ __host__ void slice_waveforms(
    Timestamp* s_timestamps, DelayInfo* s_delay_infos, char* s_values,
    Transition** data, unsigned int capacity,
    const unsigned int num_wires,
    bool* overflow_ptr
) {
    memset(s_timestamps, 0, sizeof(Timestamp) * N_STIMULI_PARALLEL * capacity);
    memset(s_delay_infos, 0, sizeof(DelayInfo) * N_STIMULI_PARALLEL * capacity);
    memset(s_values, 0, sizeof(char) * MAX_NUM_MODULE_ARGS * N_STIMULI_PARALLEL * capacity);
    unsigned int progress[MAX_NUM_MODULE_OUTPUT] = {0};

    unsigned int num_finished = 0;
    unsigned int write_stimuli_index = 0, write_transition_index = 1;

    prepare_stimuli_head(
        s_timestamps + write_stimuli_index * capacity,
        s_values + write_stimuli_index * capacity * MAX_NUM_MODULE_ARGS,
        data, num_wires, progress
    );
    for (int i = 0; i < num_wires; ++i) if (data[i][1].value == 0) num_finished++;

    while (num_finished < num_wires) {
        // find min timestamp
        Timestamp min_t = LONG_LONG_MAX;
        for (int i = 0; i < num_wires; ++i) {
            const auto& index = progress[i];
            if (OOB(index + 1, data, i)) continue;
            const auto& t = data[i][index + 1].timestamp;
            if (t < min_t) min_t = t;
        }
        assert(min_t != LONG_LONG_MAX);

        // find advancing wires
        unsigned int advancing[MAX_NUM_MODULE_ARGS], num_advancing = 0;
        for(int i = 0; i < num_wires; ++i) {
            auto& index = progress[i];
            if (OOB(index + 1, data, i)) continue;
            if (data[i][index + 1].timestamp != min_t) continue;
            advancing[num_advancing] = i; num_advancing++;
        }

        // decide where to write
        if (write_transition_index + num_advancing - 1 >= capacity) {
            write_transition_index = 1; write_stimuli_index++;
            if (write_stimuli_index >= N_STIMULI_PARALLEL) break;
            prepare_stimuli_head(
                s_timestamps + write_stimuli_index * capacity,
                s_values + write_stimuli_index * capacity * MAX_NUM_MODULE_ARGS,
                data, num_wires, progress
            );
        }
        // advance indices
        for (int i = 0; i < num_advancing; ++i) {
            auto& index = progress[advancing[i]];
            index++;
            if (OOB(index + 1, data, advancing[i])) num_finished++;
        }
        for (int i = 0; i < num_advancing; ++i) {
            s_timestamps[write_stimuli_index * capacity + write_transition_index + i] = min_t;

            const auto& advancing_arg = advancing[i];
            s_delay_infos[write_stimuli_index * capacity + write_transition_index + i].arg = advancing_arg;
            s_delay_infos[write_stimuli_index * capacity + write_transition_index + i].edge_type = get_edge_type(
                data[advancing_arg][progress[advancing_arg] - 1].value,
                data[advancing_arg][progress[advancing_arg]].value
            );
            for (int j = 0; j < num_wires; ++j) {
                const auto& transition = data[j][progress[j]];
                s_values[
                    write_stimuli_index * capacity * MAX_NUM_MODULE_ARGS
                    + (write_transition_index + i) * MAX_NUM_MODULE_ARGS
                    + j
                ] = transition.value;
            }
        }
        write_transition_index += num_advancing;
    }
    if (write_stimuli_index >= N_STIMULI_PARALLEL) *overflow_ptr = true;
}

__host__ __device__ unsigned int get_table_row_index(const char* s_input_values, unsigned int num_input) {
    unsigned int row_index = 0;
    for (unsigned int i_input = 0; i_input < num_input; ++i_input) {
        unsigned int v;
        switch (s_input_values[i_input]) {
            case '0':
                v = 0;
                break;
            case '1':
                v = 1;
                break;
            case 'x':
                v = 2;
                break;
            case 'z':
                v = 3;
                break;
        }
        row_index = (row_index << 2) + v;
    }
    return row_index;
}
__host__ __device__ void stepping_algorithm(
    const Timestamp* s_input_timestamps,
    const char* s_input_values,
    Transition** output_data,
    const ModuleSpec* module_spec,
    unsigned int capacity
) {
    for (unsigned int i = 0; i < capacity; i++) {
        if (s_input_values[i * MAX_NUM_MODULE_ARGS] == 0) break;
        auto row_index = get_table_row_index(s_input_values + i * MAX_NUM_MODULE_ARGS, module_spec->num_input);
        for (unsigned int j = 0; j < module_spec->num_output; ++j) {
            output_data[j][i].value = module_spec->table[row_index * module_spec->num_output + j];
            output_data[j][i].timestamp = s_input_timestamps[i];
        }
    }
}

__device__ Timestamp* s_input_timestamp_ptrs[N_CELL_PARALLEL];
__device__ DelayInfo* s_input_delay_info_ptrs[N_CELL_PARALLEL];
__device__ char* s_input_value_ptrs[N_CELL_PARALLEL];

__device__ void simulate_module(
    const ModuleSpec* const module_spec,
    const SDFSpec* const sdf_spec,
    Transition** const data, unsigned int capacity,
    bool* overflow_ptr
) {
    const auto& module_idx = blockIdx.x;
    auto& s_input_timestamps = s_input_timestamp_ptrs[module_idx];
    auto& s_input_delay_infos = s_input_delay_info_ptrs[module_idx];
    auto& s_input_values = s_input_value_ptrs[module_idx];

    if (threadIdx.x == 0) {
        s_input_timestamps = new Timestamp[N_STIMULI_PARALLEL * capacity];
        s_input_delay_infos = new DelayInfo[N_STIMULI_PARALLEL * capacity];
        s_input_values = new char[N_STIMULI_PARALLEL * capacity * MAX_NUM_MODULE_ARGS];
        slice_waveforms(
            s_input_timestamps, s_input_delay_infos, s_input_values,
            data, capacity,
            module_spec->num_input, overflow_ptr
        );
    }
    __syncthreads();
    assert(module_spec->num_output <= MAX_NUM_MODULE_OUTPUT);
    Transition* output_data_ptrs_for_stimuli[MAX_NUM_MODULE_OUTPUT] = { nullptr };
    unsigned stimuli_idx = threadIdx.x;
    for (unsigned int i = 0; i < module_spec->num_output; ++i) {
        output_data_ptrs_for_stimuli[i] = data[module_spec->num_input + i] + stimuli_idx * capacity;
    }

    stepping_algorithm(
        s_input_timestamps + stimuli_idx * capacity,
        s_input_values +stimuli_idx * capacity * MAX_NUM_MODULE_ARGS,
        output_data_ptrs_for_stimuli,
        module_spec,
        capacity
    );

    assert(module_spec->num_output <= MAX_NUM_MODULE_OUTPUT);
    __shared__ unsigned int lengths[N_STIMULI_PARALLEL * MAX_NUM_MODULE_OUTPUT];
    DelayInfo* delay_info_for_stimuli = s_input_delay_infos + stimuli_idx * capacity;
    compute_delay(
        output_data_ptrs_for_stimuli, capacity, delay_info_for_stimuli,
        module_spec->num_output, module_spec->num_input,
        sdf_spec, lengths + stimuli_idx * module_spec->num_output
    );

    __syncthreads();
    if (threadIdx.x == 0) {
        resolve_collisions_for_batch_stimuli(
            data, lengths, capacity,
            module_spec->num_input, module_spec->num_output
        );
        delete[] s_input_timestamps; delete[] s_input_delay_infos; delete[] s_input_values;
    }
}

__global__ void simulate_batch(BatchResource batch_resource) {
    if (blockIdx.x < batch_resource.num_modules) {
        const auto& module_spec = batch_resource.module_specs[blockIdx.x];
        const auto& sdf_spec = batch_resource.sdf_specs[blockIdx.x];
        auto* overflow_ptr = batch_resource.overflows[blockIdx.x];
        auto* module_data = &batch_resource.data_schedule[blockIdx.x * MAX_NUM_MODULE_ARGS];
        const auto& capacity = batch_resource.capacities[blockIdx.x];
        simulate_module(
            module_spec, sdf_spec, module_data, capacity, overflow_ptr
        );
    }
}

void Simulator::run() {
    cout << "| Status: Running Simulation... " << endl;

    size_t new_heap_size = N_CELL_PARALLEL * N_STIMULI_PARALLEL * INITIAL_CAPACITY * 8
            * (sizeof(Timestamp) + sizeof(DelayInfo) + sizeof(char) * MAX_NUM_MODULE_ARGS);
    cudaErrorCheck(hipDeviceSetLimit(hipLimitMallocHeapSize, new_heap_size));
    cout << "| Adjusted heap size to be " << new_heap_size  << " bytes" << endl;

    unsigned int num_layers = circuit.cell_schedule.size();
    cout << "| Total " << num_layers << " layers" << endl;

    ProgressBar progress_bar(num_layers);
    for (unsigned int i_layer = 0; i_layer < num_layers; i_layer++) {
        const auto& schedule_layer = circuit.cell_schedule[i_layer];
        stack<Cell*, std::vector<Cell*>> job_queue(schedule_layer);
        for (auto* cell : schedule_layer) cell->init();
        int session_id = 0;

        while (not job_queue.empty()) {
            unordered_set<Cell*> processing_cells;
            ResourceBuffer resource_buffer;
            for (int i = 0; i < N_CELL_PARALLEL; i++) {
                if (job_queue.empty()) break;
                auto* cell = job_queue.top(); processing_cells.insert(cell);
                cell->prepare_resource(session_id, resource_buffer);
                if (cell->finished()) job_queue.pop();
            }
            BatchResource batch_data{}; batch_data.init(resource_buffer);
            hipDeviceSynchronize(); // since async memcpy
            simulate_batch<<<N_CELL_PARALLEL, N_STIMULI_PARALLEL>>>(batch_data);
            hipDeviceSynchronize();

            for (auto* cell : processing_cells) {
                bool finished = cell->finished();
                bool overflow = cell->gather_results();
                if (finished and overflow) job_queue.push(cell);
            }
            batch_data.free();
            session_id++;
        }
        progress_bar.Progressed(i_layer + 1);
    }
    cout << endl;
}
