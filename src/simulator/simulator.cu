#include "hip/hip_runtime.h"
#include <stack>
#include <cassert>

#include "simulator/simulator.h"
#include "simulator/collision_utils.h"
#include "include/progress_bar.h"

using namespace std;


__device__ __host__ void resolve_collisions_for_batch_stimuli(
    Transition** data,
    const unsigned int* lengths,
    const unsigned int capacity,
    const unsigned int num_outputs, const unsigned int* output_indices
) {
    unsigned int stimuli_lengths[N_STIMULI_PARALLEL];
    for (int i_output = 0; i_output < num_outputs; i_output++) {
        for(int i_stimuli = 0; i_stimuli < N_STIMULI_PARALLEL; i_stimuli++) {
            stimuli_lengths[i_stimuli] = lengths[num_outputs * i_stimuli + i_output];
            assert(stimuli_lengths[i_stimuli] <= capacity);
        }
        resolve_collisions_for_batch_waveform(
            data[output_indices[i_output]], capacity,
            stimuli_lengths, N_STIMULI_PARALLEL
        );
    }
}

__device__ void simulate_module(
    const ModuleSpec* const module_spec,
    const SDFSpec* const sdf_spec,
    Transition** const data_schedule,
    const unsigned int capacity
) {
    assert(module_spec->data_schedule_size <= MAX_DATA_SCHEDULE_SIZE);
    Transition* data_ptrs_for_each_stimuli[MAX_DATA_SCHEDULE_SIZE];
    unsigned stimuli_idx = threadIdx.x;
    for (unsigned int i = 0; i < module_spec->data_schedule_size; i++) {
        data_ptrs_for_each_stimuli[i] = data_schedule[i] + stimuli_idx * capacity;
    }
    unsigned int data_schedule_idx = 0;
    for (int i = 0; i < module_spec->schedule_size; i++) {
        module_spec->gate_schedule[i](
                data_ptrs_for_each_stimuli + data_schedule_idx,
                capacity,
                module_spec->tables[i], module_spec->table_row_num[i],
                module_spec->num_inputs[i], module_spec->num_outputs[i]
        );
        data_schedule_idx += module_spec->num_inputs[i] + module_spec->num_outputs[i];
    }
    assert(module_spec->num_module_output <= MAX_NUM_MODULE_OUTPUT);
    __shared__ unsigned int lengths[N_STIMULI_PARALLEL * MAX_NUM_MODULE_OUTPUT];
    compute_delay(
            data_ptrs_for_each_stimuli, capacity,
            module_spec->output_indices, module_spec->num_module_output, module_spec->num_module_input,
            sdf_spec, lengths + stimuli_idx * module_spec->num_module_output
    );

    __syncthreads();
    if (threadIdx.x == 0) {
        resolve_collisions_for_batch_stimuli(
            data_schedule, lengths, capacity,
            module_spec->num_module_output, module_spec->output_indices
        );
    }
}

__global__ void simulate_batch(BatchResource batch_resource) {
    if (blockIdx.x < batch_resource.num_modules) {
        const auto& offset = batch_resource.data_schedule_offsets[blockIdx.x];
        const auto& module_spec = batch_resource.module_specs[blockIdx.x];
        const auto& sdf_spec = batch_resource.sdf_specs[blockIdx.x];
        auto* module_data_schedule = &batch_resource.data_schedule[offset];
        const auto& capacity = batch_resource.capacities[blockIdx.x];
        simulate_module(module_spec, sdf_spec, module_data_schedule, capacity);
    }
}

void Simulator::run() {
    cout << "| Status: Running Simulation... " << endl;

    unsigned int num_layers = circuit.cell_schedule.size();
    cout << "| Total " << num_layers << " layers" << endl;

    ProgressBar progress_bar(num_layers);
    for (unsigned int i_layer = 0; i_layer < num_layers; i_layer++) {
        const auto& schedule_layer = circuit.cell_schedule[i_layer];
        for (auto* cell : schedule_layer) cell->init();
        stack<Cell*, std::vector<Cell*>> job_queue(schedule_layer);
        int session_id = 0;

        while (not job_queue.empty()) {
            unordered_set<Cell*> processing_cells;
            ResourceBuffer resource_buffer;
            for (int i = 0; i < N_CELL_PARALLEL; i++) {
                if (job_queue.empty()) break;
                auto* cell = job_queue.top(); processing_cells.insert(cell);
                cell->prepare_resource(session_id, resource_buffer);
                if (cell->finished()) job_queue.pop();
            }
            BatchResource batch_data{}; batch_data.init(resource_buffer);
            simulate_batch<<<N_CELL_PARALLEL, N_STIMULI_PARALLEL>>>(batch_data);
            hipDeviceSynchronize();

            for (auto* cell : processing_cells) { cell->dump_result(); }
            batch_data.free();
            session_id++;
        }
        progress_bar.Progressed(i_layer + 1);
    }
    cout << endl;
}
