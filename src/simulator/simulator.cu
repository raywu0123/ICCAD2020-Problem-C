#include "hip/hip_runtime.h"
#include <stack>
#include <cassert>

#include "simulator/simulator.h"
#include "simulator/collision_utils.h"
#include "include/progress_bar.h"

using namespace std;


__device__ __host__ void resolve_collisions_for_batch_stimuli(
    Data* data,
    const unsigned int* lengths,
    const unsigned int* capacities,
    const unsigned int num_outputs, const unsigned int* output_indices
) {
    unsigned int stimuli_lengths[N_STIMULI_PARALLEL];
    for (int i_output = 0; i_output < num_outputs; i_output++) {
        for(int i_stimuli = 0; i_stimuli < N_STIMULI_PARALLEL; i_stimuli++) {
            stimuli_lengths[i_stimuli] = lengths[num_outputs * i_stimuli + i_output];
            assert(stimuli_lengths[i_stimuli] <= capacities[output_indices[i_output]]);
        }
        resolve_collisions_for_batch_waveform(
            data[output_indices[i_output]].ptr, capacities[output_indices[i_output]],
            stimuli_lengths
        );
    }
}

__device__ void init_delay_info(
    Transition** data, const unsigned int* data_schedule_args, unsigned int data_schedule_size, const unsigned int* capacities,
    unsigned int num_input
) {
    for (unsigned int i = 0; i < data_schedule_size; i++) {
        const auto& arg = data_schedule_args[i];
        if (arg >= num_input) continue;
        if (data[i][0].delay_info.edge_type == 1) continue; // use first edge type as initialize flag

        data[i][0].delay_info.edge_type = 1;
        for (unsigned int j = 1; j < capacities[i]; j++) {
            if (data[i][j].value == 0) break;
            data[i][j].delay_info.arg = arg;
            data[i][j].delay_info.edge_type = get_edge_type(data[i][j - 1].value, data[i][j].value);
        }
    }
}

__device__ void simulate_module(
    const ModuleSpec* const module_spec,
    const SDFSpec* const sdf_spec,
    Data* const data_schedule,
    bool* const overflow
) {
    assert(module_spec->data_schedule_size <= MAX_DATA_SCHEDULE_SIZE);
    __shared__ unsigned int capacities[MAX_DATA_SCHEDULE_SIZE];
    if (threadIdx.x == 0) {
        for (unsigned int i = 0; i < module_spec->data_schedule_size; i++) capacities[i] = data_schedule[i].capacity;
    }
    __syncthreads();

    Transition* data_ptrs_for_each_stimuli[MAX_DATA_SCHEDULE_SIZE];
    unsigned stimuli_idx = threadIdx.x;
    for (unsigned int i = 0; i < module_spec->data_schedule_size; i++) {
        data_ptrs_for_each_stimuli[i] = data_schedule[i].ptr + stimuli_idx * data_schedule[i].capacity;
    }

    init_delay_info(
        data_ptrs_for_each_stimuli, module_spec->data_schedule_args, module_spec->data_schedule_size, capacities,
        module_spec->num_module_input
    );
    unsigned int data_schedule_idx = 0;
    for (int i = 0; i < module_spec->schedule_size; i++) {
        module_spec->gate_schedule[i](
            data_ptrs_for_each_stimuli + data_schedule_idx,
            capacities + data_schedule_idx,
            module_spec->tables[i], module_spec->table_row_num[i],
            module_spec->num_inputs[i], module_spec->num_outputs[i],
            overflow
        );
        data_schedule_idx += module_spec->num_inputs[i] + module_spec->num_outputs[i];
    }

    assert(module_spec->num_module_output <= MAX_NUM_MODULE_OUTPUT);
    __shared__ unsigned int lengths[N_STIMULI_PARALLEL * MAX_NUM_MODULE_OUTPUT];
    compute_delay(
        data_ptrs_for_each_stimuli, capacities,
        module_spec->output_indices, module_spec->num_module_output, module_spec->num_module_input,
        sdf_spec, lengths + stimuli_idx * module_spec->num_module_output
    );

    __syncthreads();
    if (threadIdx.x == 0) {
        resolve_collisions_for_batch_stimuli(
            data_schedule, lengths, capacities,
            module_spec->num_module_output, module_spec->output_indices
        );
    }
}

__global__ void simulate_batch(BatchResource batch_resource) {
    if (blockIdx.x < batch_resource.num_modules) {
        const auto& offset = batch_resource.data_schedule_offsets[blockIdx.x];
        const auto& module_spec = batch_resource.module_specs[blockIdx.x];
        const auto& sdf_spec = batch_resource.sdf_specs[blockIdx.x];
        auto module_data_schedule = &batch_resource.data_schedule[offset];
        auto* overflow = batch_resource.overflows[blockIdx.x];
        simulate_module(module_spec, sdf_spec, module_data_schedule, overflow);
    }
}

void Simulator::run() {
    cout << "| Status: Running Simulation... " << endl;

    unsigned int num_layers = circuit.cell_schedule.size();
    cout << "| Total " << num_layers << " layers" << endl;

    ProgressBar progress_bar(num_layers);
    for (unsigned int i_layer = 0; i_layer < num_layers; i_layer++) {
        const auto& schedule_layer = circuit.cell_schedule[i_layer];
        for (auto* cell : schedule_layer) cell->init();
        stack<Cell*, std::vector<Cell*>> job_queue(schedule_layer);
        int session_id = 0;

        while (not job_queue.empty()) {
            unordered_set<Cell*> processing_cells;
            ResourceBuffer resource_buffer;
            for (int i = 0; i < N_CELL_PARALLEL; i++) {
                if (job_queue.empty()) break;
                auto* cell = job_queue.top(); processing_cells.insert(cell);
                cell->prepare_resource(session_id, resource_buffer);
                if (cell->finished()) job_queue.pop();
            }
            BatchResource batch_data{}; batch_data.init(resource_buffer);
            simulate_batch<<<N_CELL_PARALLEL, N_STIMULI_PARALLEL>>>(batch_data);
            hipDeviceSynchronize();

            for (auto* cell : processing_cells) {
                if (cell->overflow()) {
                    if (cell->finished()) job_queue.push(cell);
                    cell->handle_overflow();
                } else cell->dump_result();
            }
            batch_data.free();
            session_id++;
        }
        progress_bar.Progressed(i_layer + 1);
    }
    cout << endl;
}
