#include "hip/hip_runtime.h"
#include "simulator/simulator.h"
#include "include/progress_bar.h"

using namespace std;

extern __host__ __device__ int lookup_delay(
    Transition* wire_data,
    unsigned int input_index,
    unsigned int output_index,
    unsigned int transition_index,
    const SDFSpec* sdf_spec
) {
    char edge_type;
    if (wire_data[transition_index].value == '1' or wire_data[transition_index - 1].value == '0') {
        edge_type = '+';
    } else if (wire_data[transition_index].value == '0' or wire_data[transition_index - 1].value == '1') {
        edge_type = '-';
    } else {
        edge_type = 'x';
    }
    int delay = 0;
    for (int i_row = 0; i_row < sdf_spec->num_rows; i_row++) {
        if (sdf_spec->input_index[i_row] == input_index and sdf_spec->output_index[i_row] == output_index) {
            if (sdf_spec->edge_type[i_row] == 'x' or sdf_spec->edge_type[i_row] == edge_type) {
//                TODO assuming rising_delay == falling_delay
                delay += sdf_spec->rising_delay[i_row];
            }
        }
    }
    return delay;
}

extern __host__ __device__ void compute_delay(
    Transition** data,
    unsigned int data_schedule_size,
    unsigned int* capacities,
    unsigned int* data_schedule_indices,
    unsigned int num_inputs, unsigned int num_outputs,
    const SDFSpec* sdf_spec
) {
    auto output_indices = new unsigned int[num_outputs];
    for (int i = 0; i < data_schedule_size; i++) {
        if (num_inputs <= data_schedule_indices[i] and data_schedule_indices[i] < num_inputs + num_outputs) {
            output_indices[data_schedule_indices[i] - num_inputs] = i;
        }
    }
    auto indices = new unsigned int[data_schedule_size];
    unsigned int num_finished = 0;
    for (int i = 0; i < data_schedule_size; i++) {
        if (   data_schedule_indices[i] >= num_inputs
            or data[i][1].value == 0
            or capacities[i] == 0) num_finished++;
        indices[i] = 0;
    }
    unsigned int output_transition_index = 1;

    while(num_finished < data_schedule_size) {
        Timestamp min_timestamp = LONG_LONG_MAX;
        unsigned int min_index;
        // find min timestamp
        for (int i = 0; i < data_schedule_size; i++) {
            if (data_schedule_indices[i] >= num_inputs) continue; // not an input wire
            if (indices[i] + 1 >= capacities[i]) continue;     // out of bound
            if (data[i][indices[i] + 1].value == 0) continue;  // is padding

            const auto& transition = data[i][indices[i] + 1];
            if (transition.timestamp < min_timestamp) {
                min_timestamp = transition.timestamp;
                min_index = i;
            }
        }
        indices[min_index]++;
        for (int output_index_ = 0; output_index_ < num_outputs; output_index_++) {
            const auto& output_data = data[output_indices[output_index_]];
            if (output_transition_index >= capacities[output_indices[output_index_]]) continue;
            output_data[output_transition_index].timestamp += lookup_delay(
                data[min_index],
                data_schedule_indices[min_index], num_inputs + output_index_,
                indices[min_index],
                sdf_spec
            );
        }
        if (   indices[min_index] >= capacities[min_index] - 1
            or data[min_index][indices[min_index] + 1].value == 0) {
            num_finished++;
        }
        output_transition_index++;
    }

    delete[] output_indices;
    delete[] indices;
}

__device__ void simulate_gate_on_multiple_stimuli(
    GateFnPtr gate_fn_ptr,
    Data* data,  //(n_stimuli * capacities[i_wire], num_inputs + num_outputs)
    char* table,
    unsigned int table_row_num,
    unsigned int num_inputs, unsigned int num_outputs
) {
    unsigned int stimuli_idx = threadIdx.x;

    auto** stimuli_data = new Transition*[num_inputs + num_outputs]; // (capacities[i], num_inputs + num_outputs)
    auto* capacities  = new unsigned int[num_inputs + num_outputs];
    for (int i = 0; i < num_inputs + num_outputs; i++) {
        stimuli_data[i] = data[i].ptr + data[i].capacity * stimuli_idx;
        capacities[i] = data[i].capacity;
    }
    gate_fn_ptr(stimuli_data, capacities, table, table_row_num, num_inputs, num_outputs);

    delete[] stimuli_data;
    delete[] capacities;
}

__device__ void compute_delay_on_multiple_stimuli(
    Data* data,
    const ModuleSpec* module_spec,
    const SDFSpec* sdf_spec
) {
    unsigned int stimuli_idx = threadIdx.x;
    const auto& data_schedule_size = module_spec->data_schedule_size;
    auto** stimuli_data = new Transition*[data_schedule_size]; // (capacities[i], num_inputs + num_outputs)
    auto* capacities = new unsigned int[data_schedule_size];
    for (int i = 0; i < data_schedule_size; i++) {
        stimuli_data[i] = data[i].ptr + data[i].capacity * stimuli_idx;
        capacities[i] = data[i].capacity;
    }
    compute_delay(
        stimuli_data,
        data_schedule_size,
        capacities,
        module_spec->data_schedule_indices,
        module_spec->num_module_input, module_spec->num_module_output,
        sdf_spec
    );
    delete[] stimuli_data;
    delete[] capacities;
}

__device__ void simulate_module(
    const ModuleSpec* module_spec,
    const SDFSpec* sdf_spec,
    Data* data_schedule
) {
    unsigned int data_schedule_idx = 0;
    for (int i = 0; i < module_spec->schedule_size; i++) {
        const auto& gate_fn_ptr = module_spec->gate_schedule[i];
        const auto& table = module_spec->tables[i];
        const auto& table_row_num = module_spec->table_row_num[i];
        const auto& num_inputs = module_spec->num_inputs[i];
        const auto& num_outputs = module_spec->num_outputs[i];
        simulate_gate_on_multiple_stimuli(
            gate_fn_ptr,
            data_schedule + data_schedule_idx,
            table,
            table_row_num,
            num_inputs, num_outputs
        );
        data_schedule_idx += num_inputs + num_outputs;
    }
    compute_delay_on_multiple_stimuli(data_schedule, module_spec, sdf_spec);
}

__global__ void simulate_batch(BatchResource batch_resource) {
    if (blockIdx.x < batch_resource.num_modules) {
        const auto& offset = batch_resource.data_schedule_offsets[blockIdx.x];
        const auto& module_spec = batch_resource.module_specs[blockIdx.x];
        const auto& sdf_spec = batch_resource.sdf_specs[blockIdx.x];
        auto module_data_schedule = &batch_resource.data_schedule[offset];
        simulate_module(module_spec, sdf_spec, module_data_schedule);
    }
}

void Simulator::run() {
    cout << "Running Simulation... " << endl;

    unsigned int num_layers = circuit.cell_schedule.size();
    cout << "Total " << num_layers << " layers" << endl;

    ProgressBar progress_bar(num_layers);
    for (unsigned int i_layer = 0; i_layer < num_layers; i_layer++) {
        const auto& schedule_layer = circuit.cell_schedule[i_layer];
        for (auto* cell : schedule_layer) {
            Cell::build_bucket_index_schedule(cell->input_wires, INITIAL_CAPACITY);
        }

        int num_cells = schedule_layer.size();
        int num_finished_cells = 0;
        while (num_finished_cells < num_cells) {
            int prev_num_finished_gates = num_finished_cells;
            for (int i = 0; i < N_GATE_PARALLEL; i++) {
                const auto& cell = schedule_layer[num_finished_cells];
                if (cell->prepare_resource(resource_buffer)) {
                    num_finished_cells++;
                    if (num_finished_cells >= num_cells) break;
                }
            }
            const auto& batch_data = BatchResource{resource_buffer};
            resource_buffer.clear();
            simulate_batch<<<N_GATE_PARALLEL, N_STIMULI_PARALLEL>>>(batch_data);
            hipDeviceSynchronize();
            for (int cell_idx = prev_num_finished_gates; cell_idx < num_finished_cells; cell_idx++) {
                const auto& cell = schedule_layer[cell_idx];
                cell->finalize();
            }
        }
        progress_bar.Progressed(i_layer);
    }
    cout << endl;
}
