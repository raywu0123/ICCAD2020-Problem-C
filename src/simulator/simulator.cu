#include "hip/hip_runtime.h"
#include <stack>
#include <cassert>

#include "simulator/simulator.h"
#include "simulator/collision_utils.h"
#include "include/progress_bar.h"

using namespace std;


__device__ __host__ void resolve_collisions_for_batch_stimuli(
    Transition** data,
    const unsigned int* lengths,
    const unsigned int num_inputs, const unsigned int num_outputs
) {
//    TODO parallelize
    unsigned int stimuli_lengths[N_STIMULI_PARALLEL];
    for (int i_output = 0; i_output < num_outputs; i_output++) {
        for(int i_stimuli = 0; i_stimuli < N_STIMULI_PARALLEL; i_stimuli++) {
            stimuli_lengths[i_stimuli] = lengths[num_outputs * i_stimuli + i_output];
            assert(stimuli_lengths[i_stimuli] <= INITIAL_CAPACITY);
        }
        resolve_collisions_for_batch_waveform(data[num_inputs + i_output], stimuli_lengths, N_STIMULI_PARALLEL);
    }
}


__device__ __host__ bool OOB(unsigned int index, Transition** const data, unsigned int i) {
    return index >= N_STIMULI_PARALLEL * INITIAL_CAPACITY or data[i][index].value == 0;
}

__device__ __host__ void prepare_stimuli_head(
    Timestamp s_timestamps[INITIAL_CAPACITY],
    char s_values[INITIAL_CAPACITY][MAX_NUM_MODULE_ARGS],
    Transition** data,
    const unsigned int num_wires, unsigned int** progress_updates
) {
    s_timestamps[0] = data[0][*progress_updates[0]].timestamp;
    for (int i = 0; i < num_wires; ++i) {
        s_values[0][i] = data[i][*progress_updates[i]].value;
    }
}

__device__ __host__ void slice_waveforms(
    Timestamp s_timestamps[N_STIMULI_PARALLEL][INITIAL_CAPACITY],
    DelayInfo s_delay_infos[N_STIMULI_PARALLEL][INITIAL_CAPACITY],
    char s_values[N_STIMULI_PARALLEL][INITIAL_CAPACITY][MAX_NUM_MODULE_ARGS],
    Transition** data,
    const unsigned int num_wires, unsigned int** progress_updates
) {
    memset(s_timestamps, 0, sizeof(Timestamp) * N_STIMULI_PARALLEL * INITIAL_CAPACITY);
    memset(s_delay_infos, 0, sizeof(DelayInfo) * N_STIMULI_PARALLEL * INITIAL_CAPACITY);
    memset(s_values, 0, sizeof(char) * MAX_NUM_MODULE_ARGS * N_STIMULI_PARALLEL * INITIAL_CAPACITY);
    for(int i = 0; i < num_wires; ++i) *progress_updates[i] = 0;

    unsigned int num_finished = 0;
    unsigned int write_stimuli_index = 0, write_transition_index = 1;

    prepare_stimuli_head(
        s_timestamps[write_stimuli_index], s_values[write_stimuli_index],
        data, num_wires, progress_updates
    );
    for (int i = 0; i < num_wires; ++i) if (data[i][1].value == 0 ) num_finished++;

    while (num_finished < num_wires) {
        // find min timestamp
        Timestamp min_t = LONG_LONG_MAX;
        for (int i = 0; i < num_wires; ++i) {
            const auto& index = *progress_updates[i];
            if (OOB(index + 1, data, i)) continue;
            const auto& t = data[i][index + 1].timestamp;
            if (t < min_t) min_t = t;
        }
        assert(min_t != LONG_LONG_MAX);

        // find advancing wires
        unsigned int advancing[MAX_NUM_MODULE_ARGS], num_advancing = 0;
        for(int i = 0; i < num_wires; ++i) {
            auto& index = *progress_updates[i];
            if (OOB(index + 1, data, i)) continue;
            if (data[i][index + 1].timestamp != min_t) continue;
            advancing[num_advancing] = i; num_advancing++;
        }

        // decide where to write
        if (write_transition_index + num_advancing - 1 >= INITIAL_CAPACITY) {
            write_transition_index = 1; write_stimuli_index++;
            if (write_stimuli_index >= N_STIMULI_PARALLEL) break;
            prepare_stimuli_head(
                s_timestamps[write_stimuli_index], s_values[write_stimuli_index],
                data, num_wires, progress_updates
            );
        }
        if (write_stimuli_index >= N_STIMULI_PARALLEL) break;

        // advance indices
        for (int i = 0; i < num_advancing; ++i) {
            auto& index = *progress_updates[advancing[i]];
            index++;
            if (OOB(index + 1, data, advancing[i])) num_finished++;
        }
        for (int i = 0; i < num_advancing; ++i) {
            s_timestamps[write_stimuli_index][write_transition_index + i] = min_t;

            const auto& advancing_arg = advancing[i];
            s_delay_infos[write_stimuli_index][write_transition_index + i].arg = advancing_arg;
            s_delay_infos[write_stimuli_index][write_transition_index + i].edge_type = get_edge_type(
                data[advancing_arg][*progress_updates[advancing_arg] - 1].value,
                data[advancing_arg][*progress_updates[advancing_arg]].value
            );
            for (int j = 0; j < num_wires; ++j) {
                const auto& transition = data[j][*progress_updates[j]];
                s_values[write_stimuli_index][write_transition_index + i][j] = transition.value;
            }
        }
        write_transition_index += num_advancing;
    }
}

__host__ __device__ unsigned int get_table_row_index(
    const char s_input_values[MAX_NUM_MODULE_ARGS], unsigned int num_input
) {
    unsigned int row_index = 0;
    for (unsigned int i_input = 0; i_input < num_input; ++i_input) {
        unsigned int v;
        switch (s_input_values[i_input]) {
            case '0':
                v = 0;
                break;
            case '1':
                v = 1;
                break;
            case 'x':
                v = 2;
                break;
            case 'z':
                v = 3;
                break;
        }
        row_index = (row_index << 2) + v;
    }
    return row_index;
}
__host__ __device__ void stepping_algorithm(
    const Timestamp s_input_timestamps[INITIAL_CAPACITY],
    const char s_input_values[INITIAL_CAPACITY][MAX_NUM_MODULE_ARGS],
    Transition** output_data,
    const ModuleSpec* module_spec
) {
    for (unsigned int i = 0; i < INITIAL_CAPACITY; i++) {
        if (s_input_values[i][0] == 0) break;
        auto row_index = get_table_row_index(s_input_values[i], module_spec->num_input);
        for (unsigned int j = 0; j < module_spec->num_output; ++j) {
            output_data[j][i].value = module_spec->table[row_index * module_spec->num_output + j];
            output_data[j][i].timestamp = s_input_timestamps[i];
        }
    }
}

__device__ Timestamp sliced_input_timestamps[N_CELL_PARALLEL][N_STIMULI_PARALLEL][INITIAL_CAPACITY];
__device__ DelayInfo sliced_input_delay_infos[N_CELL_PARALLEL][N_STIMULI_PARALLEL][INITIAL_CAPACITY];
__device__ char sliced_input_values[N_CELL_PARALLEL][N_STIMULI_PARALLEL][INITIAL_CAPACITY][MAX_NUM_MODULE_ARGS];

__device__ void simulate_module(
    const ModuleSpec* const module_spec,
    const SDFSpec* const sdf_spec,
    Transition** const data,
    unsigned int** progress_updates,
    Timestamp s_input_timestamps[N_STIMULI_PARALLEL][INITIAL_CAPACITY],
    DelayInfo s_input_delay_infos[N_STIMULI_PARALLEL][INITIAL_CAPACITY],
    char s_input_values[N_STIMULI_PARALLEL][INITIAL_CAPACITY][MAX_NUM_MODULE_ARGS]
) {
    if (threadIdx.x == 0) {
        slice_waveforms(
            s_input_timestamps, s_input_delay_infos, s_input_values,
            data, module_spec->num_input, progress_updates
        );
    }
    __syncthreads();

    assert(module_spec->num_output <= MAX_NUM_MODULE_OUTPUT);
    Transition* output_data_ptrs_for_stimuli[MAX_NUM_MODULE_OUTPUT] = { nullptr };
    unsigned stimuli_idx = threadIdx.x;
    for (unsigned int i = 0; i < module_spec->num_output; ++i) {
        output_data_ptrs_for_stimuli[i] = data[module_spec->num_input + i] + stimuli_idx * INITIAL_CAPACITY;
    }

    stepping_algorithm(
        s_input_timestamps[stimuli_idx],
        s_input_values[stimuli_idx],
        output_data_ptrs_for_stimuli,
        module_spec
    );

    assert(module_spec->num_output <= MAX_NUM_MODULE_OUTPUT);
    __shared__ unsigned int lengths[N_STIMULI_PARALLEL * MAX_NUM_MODULE_OUTPUT];
    DelayInfo* delay_info_for_stimuli = s_input_delay_infos[stimuli_idx];
    compute_delay(
        output_data_ptrs_for_stimuli, delay_info_for_stimuli,
        module_spec->num_output, module_spec->num_input,
        sdf_spec, lengths + stimuli_idx * module_spec->num_output
    );

    __syncthreads();
    if (threadIdx.x == 0) {
        resolve_collisions_for_batch_stimuli(
            data, lengths,
            module_spec->num_input, module_spec->num_output
        );
    }
}

__global__ void simulate_batch(BatchResource batch_resource) {
    if (blockIdx.x < batch_resource.num_modules) {
        const auto& offset = batch_resource.data_schedule_offsets[blockIdx.x];
        const auto& module_spec = batch_resource.module_specs[blockIdx.x];
        const auto& sdf_spec = batch_resource.sdf_specs[blockIdx.x];
        auto* module_data = &batch_resource.data_schedule[offset];
        auto* progress_updates = &batch_resource.progress_updates[offset];
        simulate_module(
            module_spec, sdf_spec, module_data, progress_updates,
            sliced_input_timestamps[blockIdx.x],
            sliced_input_delay_infos[blockIdx.x],
            sliced_input_values[blockIdx.x]
        );
    }
}

void Simulator::run() {
    cout << "| Status: Running Simulation... " << endl;

    size_t new_heap_size = N_CELL_PARALLEL * N_STIMULI_PARALLEL * MAX_NUM_GATE_ARGS * sizeof(Transition) * INITIAL_CAPACITY * 4;
    cudaErrorCheck(hipDeviceSetLimit(hipLimitMallocHeapSize, new_heap_size));
    cout << "| Adjusted heap size to be " << new_heap_size  << " bytes" << endl;

    unsigned int num_layers = circuit.cell_schedule.size();
    cout << "| Total " << num_layers << " layers" << endl;

    ProgressBar progress_bar(num_layers);
    for (unsigned int i_layer = 0; i_layer < num_layers; i_layer++) {
        const auto& schedule_layer = circuit.cell_schedule[i_layer];
        stack<Cell*, std::vector<Cell*>> job_queue(schedule_layer);
        int session_id = 0;

        while (not job_queue.empty()) {
            unordered_set<Cell*> processing_cells;
            ResourceBuffer resource_buffer;
            for (int i = 0; i < N_CELL_PARALLEL; i++) {
                if (job_queue.empty()) break;
                auto* cell = job_queue.top(); job_queue.pop(); processing_cells.insert(cell);
                cell->prepare_resource(session_id, resource_buffer);
            }
            BatchResource batch_data{}; batch_data.init(resource_buffer);
            hipDeviceSynchronize(); // since async memcpy
            simulate_batch<<<N_CELL_PARALLEL, N_STIMULI_PARALLEL>>>(batch_data);
            hipDeviceSynchronize();

            for (auto* cell : processing_cells) {
                cell->gather_results();
                if (not cell->finished()) job_queue.push(cell);
            }
            batch_data.free();
            session_id++;
        }
        progress_bar.Progressed(i_layer + 1);
    }
    cout << endl;
}
