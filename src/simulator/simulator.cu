#include "hip/hip_runtime.h"
#include "simulator/simulator.h"
#include "include/progress_bar.h"

using namespace std;

__device__ void simulate_gate_on_multiple_stimuli(
    GateFnPtr gate_fn_ptr,
    Transition** data,  //(n_stimuli * capacities[i_wire], num_inputs + num_outputs)
    unsigned int* capacities,
    char* table,
    unsigned int table_row_num,
    unsigned int num_inputs, unsigned int num_outputs
) {
    unsigned int stimuli_idx = threadIdx.x;
    auto** stimuli_data = new Transition*[num_inputs + num_outputs]; // (capacities[i], num_inputs + num_outputs)
    for (int i = 0; i < num_inputs + num_outputs; i++) {
        stimuli_data[i] = data[i] + capacities[i] * stimuli_idx;
    }
    gate_fn_ptr(stimuli_data, capacities, table, table_row_num, num_inputs, num_outputs);
    delete[] stimuli_data;
}

__device__ void simulate_module(const ModuleSpec* module_spec, Transition** data_schedule, unsigned int* capacities) {
    unsigned int data_schedule_idx = 0;
    for (int i = 0; i < module_spec->schedule_size; i++) {
        const auto& gate_fn_ptr = module_spec->gate_schedule[i];
        const auto& table = module_spec->tables[i];
        const auto& table_row_num = module_spec->table_row_num[i];
        const auto& num_inputs = module_spec->num_inputs[i];
        const auto& num_outputs = module_spec->num_outputs[i];
        simulate_gate_on_multiple_stimuli(
            gate_fn_ptr,
            data_schedule + data_schedule_idx,
            capacities + data_schedule_idx,
            table,
            table_row_num,
            num_inputs, num_outputs
        );
        data_schedule_idx += num_inputs + num_outputs;
    }
}

__global__ void simulate_batch(BatchResource batch_resource) {
    if (blockIdx.x < batch_resource.num_modules) {
        const auto& offset = batch_resource.data_schedule_offsets[blockIdx.x];
        const auto& module_spec = batch_resource.module_specs[blockIdx.x];
        auto module_data_schedule = &batch_resource.data_schedule[offset];
        auto module_capacities = &batch_resource.capacities[offset];
        simulate_module(module_spec, module_data_schedule, module_capacities);
    }
};

void Simulator::run() {
    unsigned int num_batches = (int) ceil(double(input_waveforms.num_stimuli) / double(N_STIMULI_PARALLEL));
    cout << "Total " << num_batches << " batches" << endl;
    ProgressBar bar(num_batches, "Running Simulation");
    for (unsigned int i_batch = 0; i_batch < num_batches; i_batch++) {
        simulate_batch_stimuli(i_batch);
        bar.Progressed(i_batch);
    }
    cout << endl;
}

void Simulator::simulate_batch_stimuli(unsigned int& i_batch) {
    set_input(i_batch);

    for (const auto& schedule_layer : circuit.cell_schedule) {
        int n_batch_gate = ceil(double(schedule_layer.size()) / double(N_GATE_PARALLEL));
        int layer_size = schedule_layer.size();

        for (int i_batch_gate = 0; i_batch_gate < n_batch_gate; i_batch_gate++) {
            unsigned int cell_idx = i_batch_gate * N_GATE_PARALLEL;
            resource_buffer.module_specs.reserve(N_GATE_PARALLEL);
            resource_buffer.data_schedule_offsets.reserve(N_GATE_PARALLEL);
            resource_buffer.data_schedule.reserve(N_GATE_PARALLEL * 3);
            resource_buffer.capacities.reserve(N_GATE_PARALLEL * 3);
            for (; cell_idx < (i_batch_gate + 1) * N_GATE_PARALLEL and cell_idx < layer_size; cell_idx++) {
                schedule_layer[cell_idx]->prepare_resource(resource_buffer);
            }

            const auto& batch_data = get_batch_data();
            simulate_batch<<<N_GATE_PARALLEL, N_STIMULI_PARALLEL>>> (batch_data);
            // perform edge checking in the kernel
            hipDeviceSynchronize();

            for (unsigned int free_cell_idx = i_batch_gate * N_GATE_PARALLEL; free_cell_idx < cell_idx; free_cell_idx++) {
                schedule_layer[free_cell_idx]->free_resource();
//              accumulators will collect results at this stage
            }
        }
    }
}

void Simulator::set_input(unsigned int i_batch) const {
    for (int i_wire = 0; i_wire < input_waveforms.num_buckets; i_wire++) {
        const auto& bucket = input_waveforms.buckets[i_wire];
        auto& wire_ptr = bucket.wire_ptr;

        for (unsigned int i_stimuli = 0; i_stimuli < N_STIMULI_PARALLEL; i_stimuli++) {
            unsigned int global_i_stimuli = i_batch * N_STIMULI_PARALLEL + i_stimuli;
            if (global_i_stimuli >= input_waveforms.num_stimuli) break;

            wire_ptr->set_input(
                bucket.transitions,
                bucket.stimuli_edge_indices,
                global_i_stimuli
            );
        }
    }
}

BatchResource Simulator::get_batch_data() {
    BatchResource batch_resource{};
    unsigned int num_modules = resource_buffer.size();
    batch_resource.num_modules = num_modules;

    hipMalloc((void**) &batch_resource.module_specs, sizeof(ModuleSpec*) * num_modules);
    hipMalloc((void**) &batch_resource.data_schedule, sizeof(Transition*) * resource_buffer.data_schedule.size());
    hipMalloc((void**) &batch_resource.data_schedule_offsets, sizeof(unsigned int) * num_modules);
    hipMalloc((void**) &batch_resource.capacities, sizeof(unsigned int) * resource_buffer.capacities.size());

    hipMemcpy(batch_resource.module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(batch_resource.data_schedule, resource_buffer.data_schedule.data(), sizeof(Transition*) * resource_buffer.data_schedule.size(), hipMemcpyHostToDevice);
    hipMemcpy(batch_resource.data_schedule_offsets, resource_buffer.data_schedule_offsets.data(), sizeof(unsigned int) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(batch_resource.capacities, resource_buffer.capacities.data(), sizeof(unsigned int) * resource_buffer.capacities.size(), hipMemcpyHostToDevice);
    resource_buffer.clear();

    return batch_resource;
}
