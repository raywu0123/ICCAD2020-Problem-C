#include "hip/hip_runtime.h"
#include <cassert>

#include "builtin_gates.h"
#include "constants.h"


__host__ __device__ char and_logic(
    Transition** data, unsigned int num_inputs, const unsigned int* indices , const char* table, const unsigned int table_row_num
) {
    bool is_all_one = true, has_zero = false;
    for (int i = 1; i < num_inputs + 1; i++) {
        const auto& value = data[i][indices[i]].value;
        has_zero |= (value == '0');
        is_all_one &= (value == '1');
    }
    return has_zero ? '0' : (is_all_one ? '1' : 'x');
}

__host__ __device__ char or_logic(
    Transition** data, unsigned int num_inputs, const unsigned int* indices, const char* table, const unsigned int table_row_num
) {
    bool is_all_zero = true, has_one = false;
    for (int i = 1; i < num_inputs + 1; i++) {
        const auto& value = data[i][indices[i]].value;
        has_one |= (value == '1');
        is_all_zero &= (value == '0');
    }
    return has_one ? '1' : (is_all_zero ? '0' : 'x');
}
__host__ __device__ char xor_logic(
    Transition** data, unsigned int num_inputs, const unsigned int* indices, const char* table, const unsigned int table_row_num
) {
    char ret = '0';
    bool has_xz = false;
    for (int i = 1; i < num_inputs + 1; i++) {
        const auto& value = data[i][indices[i]].value;
        has_xz |= (value == 'x' or value == 'z');
        ret = (ret == value) ? '0' : '1';
    }
    return has_xz ? 'x' : ret;
}
__host__ __device__ char nand_logic(
    Transition** data, unsigned int num_inputs, const unsigned int* indices, const char* table, unsigned int table_row_num
) {
    bool is_all_one = true, has_zero = false;
    for (int i = 1; i < num_inputs + 1; i++) {
        const auto& value = data[i][indices[i]].value;
        has_zero |= (value == '0');
        is_all_one &= (value == '1');
    }
    return has_zero ? '1' : (is_all_one ? '0' : 'x');
}
__host__ __device__ char nor_logic(
    Transition** data, unsigned int num_inputs, const unsigned int* indices, const char* table, const unsigned int table_row_num
) {
    bool is_all_zero = true, has_one = false;
    for (int i = 1; i < num_inputs + 1; i++) {
        const auto& value = data[i][indices[i]].value;
        has_one |= (value == '1');
        is_all_zero &= (value == '0');
    }
    return has_one ? '0' : (is_all_zero ? '1' : 'x');
}
__host__ __device__ char xnor_logic(
    Transition** data, unsigned int num_inputs, const unsigned int* indices, const char* table, const unsigned int table_row_num
) {
    char ret = '0';
    bool has_xz = false;
    for (int i = 1; i < num_inputs + 1; i++) {
        const auto& value = data[i][indices[i]].value;
        has_xz |= (value == 'x' or value == 'z');
        ret = (ret == value) ? '0' : '1';
    }
    return has_xz ? 'x' : ret == '0' ? '1' : '0';
}
__host__ __device__ char not_logic(char v){ return (v == '0') ? '1' : ((v == '1') ? '0' : 'x'); }
__host__ __device__ char buf_logic(char v) { return  (v == 'z') ? 'x' : v; }

__host__ __device__ void merge_sort_algorithm(
    Transition** data,  // (capacity, num_inputs + num_outputs)
    const unsigned int* capacities,
    const char* table, const unsigned int table_row_num,
    const unsigned int num_inputs,
    LogicFn logic_fn,
    bool* overflow
) {
    if (data[1][0].value == 0) return;

    unsigned int indices[MAX_NUM_GATE_ARGS] = {0}; indices[0] = 1;
    assert(num_inputs + 1 <= MAX_NUM_GATE_ARGS);
    data[0][0].timestamp = data[1][0].timestamp;
    data[0][0].value = logic_fn(data, num_inputs, indices, table, table_row_num);

    unsigned int num_finished = 0;
    for (int i = 1; i < num_inputs + 1; i++) if (data[i][1].value == 0) num_finished++;

    // TODO shorter implementation
    while (num_finished < num_inputs) {
        // find min timestamp
        Timestamp min_timestamp = LONG_LONG_MAX;
        for (int i = 1; i < num_inputs + 1; i++) {
            if (indices[i] + 1 >= capacities[i] or data[i][indices[i] + 1].value == 0) continue;
            const auto& transition = data[i][indices[i] + 1];
            if (transition.timestamp < min_timestamp) min_timestamp = transition.timestamp;
        }

        // find advancing inputs, outputs
        bool advancing[MAX_NUM_GATE_ARGS] = { false };
        for (int i = 1; i < num_inputs + 1; i++) {
            if (indices[i] + 1 >= capacities[i] or data[i][indices[i] + 1].value == 0) continue;     // out of bound
            const auto& transition = data[i][indices[i] + 1];
            if (transition.timestamp == min_timestamp) {
                indices[i]++;
                advancing[i] = true;
                if (indices[i] + 1 >= capacities[i] or data[i][indices[i] + 1].value == 0) num_finished++;
            }
        }
        const char output_value = logic_fn(data, num_inputs, indices, table, table_row_num);
        for (int i = 1; i < num_inputs + 1; i++) {
            if (indices[0] >= capacities[0]) {
                *overflow = true;
                break;
            }
            if (not advancing[i]) continue;
            data[0][indices[0]].timestamp = min_timestamp; data[0][indices[0]].value = output_value;
            data[0][indices[0]].delay_info = data[i][indices[i]].delay_info; // propagate delay_info
            indices[0]++;
        }
        if (*overflow) break;
    }
}

__host__ __device__ void stepping_algorithm(Transition** data, const unsigned int* capacities, char(*logic_fn)(char)) {
    // single input doesn't overflow
    for (unsigned int i = 0; i < capacities[0]; i++) {
        data[0][i].value = data[1][i].value == 0 ? 0 : logic_fn(data[1][i].value);
        data[0][i].timestamp = data[1][i].timestamp;
        data[0][i].delay_info = data[1][i].delay_info;
    }
}
__host__ __device__ void and_gate_fn(
    Transition** data,  // (capacity, num_inputs + num_outputs)
    const unsigned int* capacities,
    const char* table, const unsigned int table_row_num,
    const unsigned int num_inputs, const unsigned int num_outputs,
    bool* overflow_ptr
) {
    merge_sort_algorithm(data, capacities, table, table_row_num, num_inputs, and_logic, overflow_ptr);
}
__host__ __device__ void or_gate_fn(
    Transition** data,  // (capacity, num_inputs + num_outputs)
    const unsigned int* capacites,
    const char* table, const unsigned int table_row_num,
    const unsigned int num_inputs, const unsigned int num_outputs,
    bool* overflow_ptr
) {
    merge_sort_algorithm(data, capacites, table, table_row_num, num_inputs, or_logic, overflow_ptr);
}
__host__ __device__ void xor_gate_fn(
    Transition** data,  // (capacity, num_inputs + num_outputs)
    const unsigned int* capacities,
    const char* table, const unsigned int table_row_num,
    const unsigned int num_inputs, const unsigned int num_outputs,
    bool* overflow_ptr
) {
    merge_sort_algorithm(data, capacities, table, table_row_num, num_inputs, xor_logic, overflow_ptr);
}
__host__ __device__ void nand_gate_fn(
    Transition** data,  // (capacity, num_inputs + num_outputs)
    const unsigned int* capacities,
    const char* table, const unsigned int table_row_num,
    unsigned int num_inputs, unsigned int num_outputs,
    bool* overflow_ptr
) {
    merge_sort_algorithm(data, capacities, table, table_row_num, num_inputs, nand_logic, overflow_ptr);
}
__host__ __device__ void nor_gate_fn(
    Transition** data,  // (capacity, num_inputs + num_outputs)
    const unsigned int* capacities,
    const char* table, const unsigned int table_row_num,
    const unsigned int num_inputs, const unsigned int num_outputs,
    bool* overflow_ptr
) {
    merge_sort_algorithm(data, capacities, table, table_row_num, num_inputs, nor_logic, overflow_ptr);
}
__host__ __device__ void xnor_gate_fn(
    Transition** data,  // (capacity, num_inputs + num_outputs)
    const unsigned int* capacities,
    const char* table, const unsigned int table_row_num,
    const unsigned int num_inputs, const unsigned int num_outputs,
    bool* overflow_ptr
) {
    merge_sort_algorithm(data, capacities, table, table_row_num, num_inputs, xnor_logic, overflow_ptr);
}
__host__ __device__ void not_gate_fn(
    Transition** data,  // (capacity, num_inputs + num_outputs)
    const unsigned int* capacities,
    const char* table, const unsigned int table_row_num,
    const unsigned int num_inputs, const unsigned int num_outputs,
    bool* overflow_ptr
) {
    stepping_algorithm(data, capacities, not_logic);
}
__host__ __device__ void buf_gate_fn(
    Transition** data,  // (capacity, num_inputs + num_outputs)
    const unsigned int* capacities,
    const char* table, const unsigned int table_row_num,
    const unsigned int num_inputs, const unsigned int num_outputs,
    bool* overflow_ptr
) {
    stepping_algorithm(data, capacities, buf_logic);
}

__host__ __device__ char primitive_logic(
    Transition** data, unsigned int num_inputs, const unsigned int* indices, const char* table, unsigned int table_row_num
) {
//    TODO optimize
    char output = 'x';  // if no matching rows, the output is x
    for (int i_table_row = 0; i_table_row < table_row_num; i_table_row++) {
        bool all_match = true;
        for (int i = 1; i < num_inputs + 1; i++) {
            auto value = data[i][indices[i]].value;
            value = (value == 'z' ? 'x' : value);  // z is treated as x
            const auto& table_value = table[i_table_row * (num_inputs + 1) + (i - 1)];
            all_match &= (table_value == '?' or table_value == value);
        }
        output = all_match ? table[i_table_row * (num_inputs + 1) + num_inputs] : output;
    }
    return output;
}
__host__ __device__ void primitive_gate_fn(
    Transition** data,
    const unsigned int* capacities,
    const char* table, const unsigned int table_row_num,
    const unsigned int num_inputs, const unsigned int num_outputs,
    bool* overflow_ptr
) {
    merge_sort_algorithm(data, capacities, table, table_row_num, num_inputs, primitive_logic, overflow_ptr);
};


__device__ GateFnPtr and_gate_fn_ptr = and_gate_fn;
__device__ GateFnPtr or_gate_fn_ptr = or_gate_fn;
__device__ GateFnPtr xor_gate_fn_ptr = xor_gate_fn;
__device__ GateFnPtr nand_gate_fn_ptr = nand_gate_fn;
__device__ GateFnPtr nor_gate_fn_ptr = nor_gate_fn;
__device__ GateFnPtr xnor_gate_fn_ptr = xnor_gate_fn;
__device__ GateFnPtr not_gate_fn_ptr = not_gate_fn;
__device__ GateFnPtr buf_gate_fn_ptr = buf_gate_fn;
__device__ GateFnPtr primitive_gate_fn_ptr = primitive_gate_fn;
