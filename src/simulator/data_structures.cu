#include "data_structures.h"

BatchResource::BatchResource(const ResourceBuffer& resource_buffer) {
    num_modules = resource_buffer.size();

    hipMalloc((void**) &module_specs, sizeof(ModuleSpec*) * num_modules);
    hipMalloc((void**) &sdf_specs, sizeof(SDFSpec*) * num_modules);
    hipMalloc((void**) &data_schedule, sizeof(Transition*) * resource_buffer.data_schedule.size());
    hipMalloc((void**) &data_schedule_offsets, sizeof(unsigned int) * num_modules);
    hipMalloc((void**) &capacities, sizeof(unsigned int) * resource_buffer.capacities.size());

    hipMemcpy(module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(sdf_specs, resource_buffer.sdf_specs.data(), sizeof(SDFSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(data_schedule, resource_buffer.data_schedule.data(), sizeof(Transition*) * resource_buffer.data_schedule.size(), hipMemcpyHostToDevice);
    hipMemcpy(data_schedule_offsets, resource_buffer.data_schedule_offsets.data(), sizeof(unsigned int) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(capacities, resource_buffer.capacities.data(), sizeof(unsigned int) * resource_buffer.capacities.size(), hipMemcpyHostToDevice);
}

BatchResource::~BatchResource() {
    hipFree(module_specs);
    hipFree(sdf_specs);
    hipFree(data_schedule);
    hipFree(data_schedule_offsets);
    hipFree(capacities);
}
