#include <iostream>

#include "data_structures.h"

std::ostream& operator<< (std::ostream& os, const Transition& transition) {
    os << "(" << transition.timestamp << ", " << transition.value << ")";
    return os;
}

void BatchResource::init(const ResourceBuffer& resource_buffer) {
    num_modules = resource_buffer.size();

    hipMalloc((void**) &module_specs, sizeof(ModuleSpec*) * num_modules);
    hipMalloc((void**) &sdf_specs, sizeof(SDFSpec*) * num_modules);
    hipMalloc((void**) &data_schedule, sizeof(Transition*) * resource_buffer.data_schedule.size());
    hipMalloc((void**) &data_schedule_offsets, sizeof(unsigned int) * num_modules);
    hipMalloc((void**) &capacities, sizeof(unsigned int) * num_modules);
    hipMalloc((void**) &verbose, sizeof(int) * num_modules);

    hipMemcpy(module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(sdf_specs, resource_buffer.sdf_specs.data(), sizeof(SDFSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(data_schedule, resource_buffer.data_schedule.data(), sizeof(Transition*) * resource_buffer.data_schedule.size(), hipMemcpyHostToDevice);
    hipMemcpy(data_schedule_offsets, resource_buffer.data_schedule_offsets.data(), sizeof(unsigned int) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(capacities, resource_buffer.capacities.data(), sizeof(unsigned int) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(verbose, resource_buffer.verbose.data(), sizeof(int) * num_modules, hipMemcpyHostToDevice);
}

void BatchResource::free() const {
    hipFree(module_specs);
    hipFree(sdf_specs);
    hipFree(data_schedule);
    hipFree(data_schedule_offsets);
    hipFree(capacities);
    hipFree(verbose);
}
