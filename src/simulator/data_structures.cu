#include <iostream>
#include "data_structures.h"

std::ostream& operator<< (std::ostream& os, const Transition& transition) {
    os << "(" << transition.timestamp << ", " << transition.value << ")";
    return os;
}

void BatchResource::init(const ResourceBuffer& resource_buffer) {
    num_modules = resource_buffer.size();

    hipMalloc((void**) &overflows, sizeof(bool*) * num_modules);
    hipMalloc((void**) &module_specs, sizeof(ModuleSpec*) * num_modules);
    hipMalloc((void**) &sdf_specs, sizeof(SDFSpec*) * num_modules);
    hipMalloc((void**) &data_schedule, sizeof(Data) * resource_buffer.data_schedule.size());
    hipMalloc((void**) &data_schedule_offsets, sizeof(unsigned int) * num_modules);

    hipMemcpy(overflows, resource_buffer.overflows.data(), sizeof(bool*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(sdf_specs, resource_buffer.sdf_specs.data(), sizeof(SDFSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(data_schedule, resource_buffer.data_schedule.data(), sizeof(Data) * resource_buffer.data_schedule.size(), hipMemcpyHostToDevice);
    hipMemcpy(data_schedule_offsets, resource_buffer.data_schedule_offsets.data(), sizeof(unsigned int) * num_modules, hipMemcpyHostToDevice);
}

void BatchResource::free() const {
    hipFree(overflows);
    hipFree(module_specs);
    hipFree(sdf_specs);
    hipFree(data_schedule);
    hipFree(data_schedule_offsets);
}
