#include <iostream>

#include "constants.h"
#include "data_structures.h"

using namespace std;

std::ostream& operator<< (std::ostream& os, const Transition& transition) {
    os << "(" << transition.timestamp << ", " << transition.value << ")";
    return os;
}

void BatchResource::init(const ResourceBuffer& resource_buffer) {
    num_modules = resource_buffer.size;

    hipMalloc((void**) &overflows, sizeof(bool*) * num_modules);
    hipMalloc((void**) &capacities, sizeof(unsigned int) * num_modules);
    hipMalloc((void**) &module_specs, sizeof(ModuleSpec*) * num_modules);
    hipMalloc((void**) &sdf_specs, sizeof(SDFSpec*) * num_modules);
    hipMalloc((void**) &data_schedule, sizeof(Data) * resource_buffer.data_schedule.size());

    hipMemcpy(overflows, resource_buffer.overflows.data(), sizeof(bool*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(capacities, resource_buffer.capacities.data(), sizeof(unsigned int) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(sdf_specs, resource_buffer.sdf_specs.data(), sizeof(SDFSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(data_schedule, resource_buffer.data_schedule.data(), sizeof(Data) * resource_buffer.data_schedule.size(), hipMemcpyHostToDevice);
}

void BatchResource::free() const {
    hipFree(overflows);
    hipFree(capacities);
    hipFree(module_specs);
    hipFree(sdf_specs);
    hipFree(data_schedule);
}

ResourceBuffer::ResourceBuffer() {
    overflows.reserve(N_CELL_PARALLEL);
    capacities.reserve(N_CELL_PARALLEL);
    module_specs.reserve(N_CELL_PARALLEL);
    sdf_specs.reserve(N_CELL_PARALLEL);
    data_schedule.reserve(N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
}

void ResourceBuffer::finish_module() {
    size++;
    data_schedule.resize(size * MAX_NUM_MODULE_ARGS);
}
