#include <iostream>

#include "constants.h"
#include "data_structures.h"

using namespace std;

Values raw_to_enum(char v) {
    switch (v) {
        case '0':
            return Values::ZERO;
        case '1':
            return Values::ONE;
        case 'x':
        case 'X':
            return Values::X;
        case 'z':
        case 'Z':
            return Values::Z;
        default:
            return Values::PAD;
    }
}

char enum_to_raw(Values v) {
    switch (v) {
        case Values::ZERO:
            return '0';
        case Values::ONE:
            return '1';
        case Values::X:
            return 'x';
        case Values::Z:
            return 'z';
        default:
            return '_';
    }
}

inline std::ostream& operator<< (std::ostream& os, const Values& v) {
    os << enum_to_raw(v);
    return os;
}

std::ostream& operator<< (std::ostream& os, const Transition& transition) {
    os << "(" << transition.timestamp << ", " << transition.value << ")";
    return os;
}

void BatchResource::init(const ResourceBuffer& resource_buffer) {
    num_modules = resource_buffer.size;

    hipMalloc((void**) &overflows, sizeof(bool*) * num_modules);
    hipMalloc((void**) &capacities, sizeof(unsigned int) * num_modules);
    hipMalloc((void**) &module_specs, sizeof(ModuleSpec*) * num_modules);
    hipMalloc((void**) &sdf_specs, sizeof(SDFSpec*) * num_modules);
    hipMalloc((void**) &data_schedule, sizeof(Data) * resource_buffer.data_schedule.size());

    hipMemcpy(overflows, resource_buffer.overflows.data(), sizeof(bool*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(capacities, resource_buffer.capacities.data(), sizeof(unsigned int) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(sdf_specs, resource_buffer.sdf_specs.data(), sizeof(SDFSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(data_schedule, resource_buffer.data_schedule.data(), sizeof(Data) * resource_buffer.data_schedule.size(), hipMemcpyHostToDevice);
}

void BatchResource::free() const {
    hipFree(overflows);
    hipFree(capacities);
    hipFree(module_specs);
    hipFree(sdf_specs);
    hipFree(data_schedule);
}

ResourceBuffer::ResourceBuffer() {
    overflows.reserve(N_CELL_PARALLEL);
    capacities.reserve(N_CELL_PARALLEL);
    module_specs.reserve(N_CELL_PARALLEL);
    sdf_specs.reserve(N_CELL_PARALLEL);
    data_schedule.reserve(N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
}

void ResourceBuffer::finish_module() {
    size++;
    data_schedule.resize(size * MAX_NUM_MODULE_ARGS);
}
