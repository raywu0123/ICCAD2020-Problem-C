#include <iostream>

#include "constants.h"
#include "data_structures.h"

std::ostream& operator<< (std::ostream& os, const Transition& transition) {
    os << "(" << transition.timestamp << ", " << transition.value << ")";
    return os;
}

void BatchResource::init(const ResourceBuffer& resource_buffer) {
    num_modules = resource_buffer.size();

    hipMalloc((void**) &module_specs, sizeof(ModuleSpec*) * num_modules);
    hipMalloc((void**) &sdf_specs, sizeof(SDFSpec*) * num_modules);
    hipMalloc((void**) &data_schedule, sizeof(Transition*) * resource_buffer.data_schedule.size());
    hipMalloc((void**) &data_schedule_offsets, sizeof(unsigned int) * num_modules);

    hipMemcpy(module_specs, resource_buffer.module_specs.data(), sizeof(ModuleSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(sdf_specs, resource_buffer.sdf_specs.data(), sizeof(SDFSpec*) * num_modules, hipMemcpyHostToDevice);
    hipMemcpy(data_schedule, resource_buffer.data_schedule.data(), sizeof(Transition*) * resource_buffer.data_schedule.size(), hipMemcpyHostToDevice);
    hipMemcpy(data_schedule_offsets, resource_buffer.data_schedule_offsets.data(), sizeof(unsigned int) * num_modules, hipMemcpyHostToDevice);
}

void BatchResource::free() const {
    hipFree(module_specs);
    hipFree(sdf_specs);
    hipFree(data_schedule);
    hipFree(data_schedule_offsets);
}

ResourceBuffer::ResourceBuffer() {
    module_specs.reserve(N_CELL_PARALLEL);
    sdf_specs.reserve(N_CELL_PARALLEL);
    data_schedule_offsets.reserve(N_CELL_PARALLEL);
    data_schedule.reserve(N_CELL_PARALLEL * MAX_NUM_MODULE_ARGS);
}

int ResourceBuffer::size() const {
    return module_specs.size();
}
