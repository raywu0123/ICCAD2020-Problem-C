#include <iostream>
#include <fstream>
#include "module_registry.h"
#include "builtin_gates.h"

using namespace std;


ModuleRegistry::ModuleRegistry() {
    register_primitives();
}
//TODO destructor: free module_spec and char table

void ModuleRegistry::summary() const {
    cout << "Module Registry Summary: " << endl;
    cout << "Num gates: " << name_to_gate.size() << endl;
    cout << "Num UDP: " << name_to_table.size() << endl;
    cout << "Num modules: " << name_to_module_spec.size() << endl;
    cout << endl;
}


void ModuleRegistry::read_file(ifstream& fin) {
    int num_primitive_cells, num_modules;
    fin >> num_primitive_cells >> num_modules;

    for (int i = 0; i < num_primitive_cells; i++) {
        read_vlib_primitive(fin);
    }
    for (int i = 0; i < num_modules; i++) {
        read_vlib_module(fin);
    }
}

void ModuleRegistry::read_vlib_primitive(ifstream& fin) {
    StdCellDeclare declares;
    string name = read_vlib_common(fin, declares);
    vector<string> table;
    read_vlib_table(fin, table);

    register_user_defined_primitive(name, table, declares);
}

void ModuleRegistry::read_vlib_table(ifstream& fin, vector<string>& table) {
    int num_rows;
    fin >> num_rows;
    string row;
    for (int i = 0; i < num_rows; i++) {
        fin >> row;
        table.push_back(row);
    }
}

void ModuleRegistry::read_vlib_module(ifstream& fin) {
    StdCellDeclare declares;
    string name = read_vlib_common(fin, declares);
    int num_submodules;

    fin >> num_submodules;
    vector<SubmoduleSpec> submodule_specs;
    for (int i = 0; i < num_submodules; i++) {
        string s;
        int num_args;
        SubmoduleSpec submodule_spec;
        fin >> s >> submodule_spec.type >> submodule_spec.name >> num_args;
        if (num_args < 2)
            throw runtime_error("Less than two args to submodule " + submodule_spec.name + " in " + name + '\n');

        for (int i_arg = 0; i_arg < num_args; i_arg++) {
            unsigned int arg_index;
            fin >> arg_index;
            submodule_spec.args.push_back(arg_index);
        }
        submodule_specs.push_back(submodule_spec);
    }
    register_module(name, submodule_specs, declares);
}

string ModuleRegistry::read_vlib_common(ifstream& fin, StdCellDeclare& declares) {
    string name;
    fin >> name;
    for (auto& arg_bucket : declares.buckets) {
        int num_args;
        unsigned int arg_index;
        string s;
        fin >> s >> num_args;
        for (int i = 0; i < num_args; i++){
            fin >> arg_index;
            arg_bucket.push_back(arg_index);
        }
    }
    return name;
}

extern __device__ GateFnPtr and_gate_fn_ptr;
extern __device__ GateFnPtr or_gate_fn_ptr;
extern __device__ GateFnPtr xor_gate_fn_ptr;
extern __device__ GateFnPtr nand_gate_fn_ptr;
extern __device__ GateFnPtr nor_gate_fn_ptr;
extern __device__ GateFnPtr xnor_gate_fn_ptr;
extern __device__ GateFnPtr not_gate_fn_ptr;
extern __device__ GateFnPtr buf_gate_fn_ptr;
void ModuleRegistry::register_primitives() {
    GateFnPtr host_and_gate_fn_ptr;
    GateFnPtr host_or_gate_fn_ptr;
    GateFnPtr host_xor_gate_fn_ptr;
    GateFnPtr host_nand_gate_fn_ptr;
    GateFnPtr host_nor_gate_fn_ptr;
    GateFnPtr host_xnor_gate_fn_ptr;
    GateFnPtr host_not_gate_fn_ptr;
    GateFnPtr host_buf_gate_fn_ptr;
    hipMemcpyFromSymbol(&host_and_gate_fn_ptr, HIP_SYMBOL(and_gate_fn_ptr), sizeof(GateFnPtr));
    hipMemcpyFromSymbol(&host_or_gate_fn_ptr, HIP_SYMBOL(or_gate_fn_ptr), sizeof(GateFnPtr));
    hipMemcpyFromSymbol(&host_xor_gate_fn_ptr, HIP_SYMBOL(xor_gate_fn_ptr), sizeof(GateFnPtr));
    hipMemcpyFromSymbol(&host_nand_gate_fn_ptr, HIP_SYMBOL(nand_gate_fn_ptr), sizeof(GateFnPtr));
    hipMemcpyFromSymbol(&host_nor_gate_fn_ptr, HIP_SYMBOL(nor_gate_fn_ptr), sizeof(GateFnPtr));
    hipMemcpyFromSymbol(&host_xnor_gate_fn_ptr, HIP_SYMBOL(xnor_gate_fn_ptr), sizeof(GateFnPtr));
    hipMemcpyFromSymbol(&host_not_gate_fn_ptr, HIP_SYMBOL(not_gate_fn_ptr), sizeof(GateFnPtr));
    hipMemcpyFromSymbol(&host_buf_gate_fn_ptr, HIP_SYMBOL(buf_gate_fn_ptr), sizeof(GateFnPtr));
    name_to_gate["and"] = host_and_gate_fn_ptr;
    name_to_gate["or"] = host_or_gate_fn_ptr;
    name_to_gate["xor"] = host_xor_gate_fn_ptr;
    name_to_gate["nand"] = host_nand_gate_fn_ptr;
    name_to_gate["nor"] = host_nor_gate_fn_ptr;
    name_to_gate["xnor"] = host_xnor_gate_fn_ptr;
    name_to_gate["not"] = host_not_gate_fn_ptr;
    name_to_gate["buf"] = host_buf_gate_fn_ptr;
}

void ModuleRegistry::register_user_defined_primitive(
    const string &name,
    const vector<string>& table,
    const StdCellDeclare& declares
) {
    if (name_to_gate.find(name) != name_to_gate.end()) {
        throw runtime_error("Primitive names shadows gate name: " + name + "\n");
    }
    if (table.empty()) {
        throw runtime_error("Emtpy table for UDP: " + name + "\n");
    }
    if(name_to_declares.find(name) != name_to_declares.end()) {
        throw runtime_error("Duplicate modules: " + name + "\n");
    }
    name_to_declares[name] = declares;

    int row_size = table[0].size();
    Table table_struct;
    table_struct.num_rows = table.size();
    table_struct.table = new char[table_struct.num_rows * row_size]; // temporary
//    TODO move char_table to constant memory
    for(int i = 0; i < table_struct.num_rows; i++) {
        for (int j = 0; j < row_size; j++) {
            table_struct.table[i * row_size + j] = table[i][j];
        }
    }
    char* device_char_table;
    hipMalloc((void**) &device_char_table, table_struct.num_rows * row_size);
    hipMemcpy(device_char_table, table_struct.table, sizeof(table_struct.num_rows) * row_size, hipMemcpyHostToDevice);
    delete[] table_struct.table;
    table_struct.table = device_char_table;
    name_to_table[name] = table_struct;
}

GateFnPtr ModuleRegistry::get_gate_fn(const string &name, char*& table, unsigned int& table_row_num) const {
    const auto& gate_it = name_to_gate.find(name);
    if (gate_it != name_to_gate.end()) return gate_it->second;
    const auto& table_it = name_to_table.find(name);
    if (table_it != name_to_table.end()) {
        table = table_it->second.table;
        table_row_num = table_it->second.num_rows;
        return nullptr;
    }

    throw runtime_error("Gate " + name + " not found.\n");
}

void ModuleRegistry::register_module(
    const string& name,
    const vector<SubmoduleSpec>& submodules,
    const StdCellDeclare& declares
) {
    if(name_to_module_spec.find(name) != name_to_module_spec.end()) {
        throw runtime_error("Duplicate modules: " + name + "\n");
    }
    if(name_to_submodule_specs.find(name) != name_to_submodule_specs.end()) {
        throw runtime_error("Duplicate modules: " + name + "\n");
    }
    if(name_to_declares.find(name) != name_to_declares.end()) {
        throw runtime_error("Duplicate modules: " + name + "\n");
    }
    name_to_submodule_specs[name] = submodules;
    name_to_declares[name] = declares;

//    temporary
//    TODO move module_spec to constant memory
    if (submodules.empty())
        throw runtime_error("Empty module " + name + "\n");
    vector<unsigned int> data_schedule_indices;
    for (const auto& submodule_spec : submodules) {
        data_schedule_indices.insert(
            data_schedule_indices.end(),
            submodule_spec.args.begin(),
            submodule_spec.args.end()
        );
    }

    vector<GateFnPtr> gate_schedule;
    vector<char*> tables;
    vector<unsigned int> table_row_nums, num_inputs, num_outputs;
    unsigned int schedule_size = submodules.size();
    for (int i = 0; i < schedule_size; i++) {
        char* table;
        unsigned int table_row_num;
        gate_schedule.push_back(get_gate_fn(submodules[i].type, table, table_row_num));
        tables.push_back(table);
        table_row_nums.push_back(table_row_num);
        num_outputs.push_back(1);
        num_inputs.push_back(submodules[i].args.size() - 1);
    }
    ModuleSpec device_module_spec_{};
    device_module_spec_.schedule_size = schedule_size;
    device_module_spec_.data_schedule_size = data_schedule_indices.size();
    hipMalloc((void**) &device_module_spec_.data_schedule_indices, sizeof(unsigned int) * data_schedule_indices.size());
    hipMemcpy(device_module_spec_.data_schedule_indices, data_schedule_indices.data(), sizeof(unsigned int) * data_schedule_indices.size(), hipMemcpyHostToDevice);
    hipMalloc((void**) &device_module_spec_.gate_schedule, sizeof(GateFnPtr) * schedule_size);
    hipMemcpy(device_module_spec_.gate_schedule, gate_schedule.data(), sizeof(GateFnPtr) * schedule_size, hipMemcpyHostToDevice);
    hipMalloc((void**) &device_module_spec_.tables, sizeof(char*) * schedule_size);
    hipMemcpy(device_module_spec_.tables, tables.data(), sizeof(char*) * schedule_size, hipMemcpyHostToDevice);
    hipMalloc((void**) &device_module_spec_.table_row_num, sizeof(unsigned int) * schedule_size);
    hipMemcpy(device_module_spec_.table_row_num, table_row_nums.data(), sizeof(unsigned int) * schedule_size, hipMemcpyHostToDevice);
    hipMalloc((void**) &device_module_spec_.num_inputs, sizeof(unsigned int) * schedule_size);
    hipMemcpy(device_module_spec_.num_inputs, num_inputs.data(), sizeof(unsigned int) * schedule_size, hipMemcpyHostToDevice);
    hipMalloc((void**) &device_module_spec_.num_outputs, sizeof(unsigned int) * schedule_size);
    hipMemcpy(device_module_spec_.num_outputs, num_outputs.data(), sizeof(unsigned int) * schedule_size, hipMemcpyHostToDevice);

    ModuleSpec* device_module_spec;
    hipMalloc((void**) &device_module_spec, sizeof(ModuleSpec));
    hipMemcpy(device_module_spec, &device_module_spec_, sizeof(ModuleSpec), hipMemcpyHostToDevice);
    name_to_module_spec[name] = device_module_spec;
}

const ModuleSpec* ModuleRegistry::get_module_spec(const string &cell_type) const {
    const auto& it = name_to_module_spec.find(cell_type);
    if (it == name_to_module_spec.end())
        throw runtime_error("ModuleSpec for type " + cell_type + " not found.");
    return it->second;
}

const vector<SubmoduleSpec>* ModuleRegistry::get_submodule_specs(const string &cell_type) const {
    const auto& it = name_to_submodule_specs.find(cell_type);
    if (it == name_to_submodule_specs.end())
        throw runtime_error("SubmoduleSpecs for type " + cell_type + " not found.");
    return &it->second;
}

const StdCellDeclare* ModuleRegistry::get_module_declare(const string &cell_type) const {
    const auto& it = name_to_declares.find(cell_type);
    if (it == name_to_declares.end())
        throw runtime_error("Declares for type " + cell_type + " not found.");
    return &it->second;
}
