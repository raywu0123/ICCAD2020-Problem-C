#include <iostream>

#include "memory_manager.h"
#include "utils.h"

using namespace std;

Data MemoryManager::alloc(size_t size) {
//    TODO buddy tree
    Transition* t;
    cudaErrorCheck(hipMalloc((void**) &t, sizeof(Transition) * size));
    unsigned int* i;
    cudaErrorCheck(hipMalloc((void**) &i, sizeof(unsigned int)));
    return {t, i};
}

void MemoryManager::free(Data d) {
    cudaErrorCheck(hipFree(d.transitions));
    cudaErrorCheck(hipFree(d.size));
}
