#include "memory_manager.h"

void MemoryManager::init() {

}

Transition* MemoryManager::alloc(size_t size) {
//    TODO buddy tree
    Transition* p;
    hipMalloc((void**) &p, sizeof(Transition) * size);
    hipMemset(&p, 0, sizeof(Transition) * size);
    return p;
}

void MemoryManager::free(Transition* p) {
    hipFree(p);
}
