#include <iostream>

#include "memory_manager.h"

using namespace std;

Transition* MemoryManager::alloc(size_t size) {
//    TODO buddy tree
    Transition* p;
    auto status = hipMalloc((void**) &p, sizeof(Transition) * size);
    if (status != hipSuccess) throw std::runtime_error(hipGetErrorString(status));
    return p;
}

void MemoryManager::free(Transition* p) {
    auto status = hipFree(p);
    if (status != hipSuccess) throw std::runtime_error(hipGetErrorString(status));
}
