#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <functional>
#include <cassert>

#include "vcd_reader.h"

using namespace std;


void VCDReader::ignore_vcd_header(ifstream& fin) {
    string s;
    do{ fin >> s; } while(s != "$timescale");
}

InputInfo VCDReader::read_input_info() {
    ignore_vcd_header(fin);
    string s;

    pair<int, string> timescale_pair;
    fin >> timescale_pair.first >> timescale_pair.second >> s;
    InputInfo info(timescale_pair);

    while (s != "$var") {
        fin >> s;
        if (s == "$scope") {
            fin >> s >> s;
            info.scopes.push_back(s);
        }
    }
    return info;
}

void VCDReader::summary() const {
    cout << "Summary of Input Waveforms" << endl;
    cout << "Num dumps: " << n_dump << endl;
    cout << endl;
}

void VCDReader::read_input_waveforms(Circuit& circuit) {
    cout << "| STATUS: Reading Input VCD file..." << endl;
    read_vars();
    get_buckets(circuit);
    read_dump();
    fin.close();
}

void VCDReader::read_vars() {
    string s;
    do {
        string token, id;
        unsigned int n_bits;
        fin >> s >> n_bits>> token >> id;
        char c;
        BitWidth bitwidth = {0, 0};
        if (n_bits > 1) fin >> c >> bitwidth.first >> c >> bitwidth.second >> c;
        token_to_wire.emplace(token, TokenInfo{id, bitwidth, 0});
        fin >> s >> s;
    } while (s.find("$var") == 0);

    while (s.find("$dumpvars") == string::npos) { getline(fin, s); }
}

void VCDReader::get_buckets(Circuit& circuit) {
    for (auto& it : token_to_wire) {
        auto& token_info = it.second;
        token_info.bucket_index = buckets.size();
        const auto& bitwidth = token_info.bitwidth;
        int step = bitwidth.first > bitwidth.second ? -1 : 1;
        for (int bit_index = bitwidth.first; bit_index != bitwidth.second + step; bit_index += step) {
            // buckets in MSB -> LSB order
            const auto& wire = circuit.get_wire(Wirekey{token_info.wire_name, bit_index});
            buckets.push_back(&wire->bucket);
        }
    }
}

void VCDReader::read_dump() {
    char c;
    fin >> c;
    while (not fin.eof()) {
        Timestamp t;
        fin >> t;
        read_single_time_dump(t);
        n_dump++;
    }
}

void VCDReader::read_single_time_dump(const Timestamp& timestamp) {
    char c;
    fin >> c;
    while (c != '#' and c != EOF and not fin.eof()) {
        string token;
        if (c == 'b') {
            string value;
            fin >> value >> token;
            emplace_transition(token, timestamp, value);
        } else {
            fin >> token;
            emplace_transition(token, timestamp, c);
        }
        fin >> c;
    }
}

void VCDReader::emplace_transition(const string& token, const Timestamp& timestamp, const string& value) {
    const auto& it = token_to_wire.find(token);
    if (it == token_to_wire.end())
        throw runtime_error("Token " + token + " not found at t = " + to_string(timestamp) + "\n");
    const auto& token_info = it->second;

    const auto& bitwidth = token_info.bitwidth;
    unsigned int bit_range = abs(bitwidth.first - bitwidth.second) + 1;
    const auto& value_size = value.size();
    assert(bit_range >= value_size);
    unsigned pad_size = bit_range - value_size;
    for (unsigned int bit_index = 0; bit_index < pad_size; ++bit_index) {
        char bit_value = value[0] == '1' ? '0' : value[0];
        buckets[token_info.bucket_index + bit_index]->emplace_transition(timestamp, bit_value);
    }
    for (unsigned int bit_index = pad_size; bit_index < bit_range; ++bit_index) {
        const char& bit_value = value[bit_index - pad_size];
        buckets[token_info.bucket_index + bit_index]->emplace_transition(timestamp, bit_value);
    }
}

void VCDReader::emplace_transition(const string& token, const Timestamp& timestamp, const char& value) {
    const auto& it = token_to_wire.find(token);
    if (it == token_to_wire.end())
        throw runtime_error("Token " + token + " not found at t = " + to_string(timestamp) + "\n");
    const auto& token_info = it->second;
    auto* bucket = buckets[token_info.bucket_index];
    bucket->emplace_transition(timestamp, value);
}
